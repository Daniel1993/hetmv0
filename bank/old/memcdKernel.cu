#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"
#include <time.h>

#include "bankKernel.cuh"
#include "bitmap.h"

// __constant__ __device__ long dev_basePoint; // TODO: check how global variables are linked in CUDA

// TODO: implement

#ifdef PR_MAX_RWSET_SIZE
#undef PR_MAX_RWSET_SIZE NUMBER_WAYS
#endif

#ifdef PR_ARGS_S_EXT
#undef PR_ARGS_S_EXT
#endif

#if CMP_TYPE == CMP_EXPLICIT
/* TODO: need logPos (do every thread commit the same number of transactions?) */
#define HeTM_GPU_log_explicit_s \
	unsigned explicitLogBlock; \
	unsigned *explicitLogOffThr; \
//
#define HeTM_GPU_log_explicit_prepare \
	size_t explicitLogCounter = PR_threadNum*PR_blockNum; \
	GPU_log->explicitLogOffThr = (unsigned*)memman_ad_hoc_alloc(NULL, NULL, explicitLogCounter*sizeof(unsigned)); \
	memman_ad_hoc_zero(NULL); \
	GPU_log->explicitLogBlock = EXPLICIT_LOG_BLOCK; \
//
#define HeTM_GPU_log_explicit_before_reads \
	int tid_ = blockIdx.x*blockDim.x + threadIdx.x; \
	int explicitLogOffset = tid_ * GPU_log->explicitLogBlock; \
//
#define HeTM_GPU_log_explicit_after_reads \
	GPU_log->explicitLogOffThr[tid_] += BANK_NB_TRANSFERS; \
//
#define HeTM_GPU_log_explicit_teardown \
	memman_ad_hoc_free(NULL); \
//
#elif CMP_TYPE == CMP_COMPRESSED
#define HeTM_GPU_log_explicit_s                  /* empty */
#define HeTM_GPU_log_explicit_prepare            /* empty */
#define HeTM_GPU_log_explicit_before_reads       /* empty */
#define HeTM_GPU_log_explicit_after_reads        /* empty */
#define HeTM_GPU_log_explicit_teardown           /* empty */
#else
// error or disabled
#define HeTM_GPU_log_explicit_s                  /* empty */
#define HeTM_GPU_log_explicit_prepare            /* empty */
#define HeTM_GPU_log_explicit_before_reads       /* empty */
#define HeTM_GPU_log_explicit_after_reads        /* empty */
#define HeTM_GPU_log_explicit_teardown           /* empty */
#endif

#define PR_ARGS_S_EXT \
	typedef struct { \
		void *dev_rset; \
		int *onIntersect; \
		hiprandState *state; \
		long CPUAccountsBasePtr; \
		HeTM_GPU_log_explicit_s /* Explicit log only */ \
	} HeTM_GPU_log_s \

#ifdef PR_DEV_BUFF_S_EXT
#undef PR_DEV_BUFF_S_EXT
#endif

#define PR_DEV_BUFF_S_EXT \
	typedef struct { \
		HeTM_GPU_log_s gpuLog; \
		hiprandState *state; \
	} HeTM_GPU_dbuf_log_s \

#ifdef PR_AFTER_PREPARE_EXT
#undef PR_AFTER_PREPARE_EXT
#endif

#define PR_AFTER_PREPARE_EXT(args) ({ \
	HeTM_GPU_log_s *GPU_log; \
	memman_alloc_dual("HeTM_gpuLog", sizeof(HeTM_GPU_log_s), MEMMAN_THRLC); \
	GPU_log = (HeTM_GPU_log_s*)memman_get_cpu(NULL); \
	/* TODO: explicit log only */ \
	HeTM_GPU_log_explicit_prepare \
	/* ---------------------- */ \
	GPU_log->CPUAccountsBasePtr = (long)args->host.inBuf; \
	memman_select("HeTM_dev_rset"); \
	GPU_log->dev_rset = memman_get_gpu(NULL); \
	memman_select("Stats_OnIntersect"); \
	GPU_log->onIntersect = (int*)memman_get_gpu(NULL); \
	GPU_log->state = (hiprandState*)parsedData.cd->devStates; \
	args->host.pr_args_ext = (void*)GPU_log; \
	memman_select("HeTM_gpuLog"); \
	args->dev.pr_args_ext = memman_get_gpu(NULL); \
	memman_cpy_to_gpu(NULL); \
}) \
//
// TODO: no need of coping back the log

#ifdef PR_AFTER_VAL_LOCKS_EXT
#undef PR_AFTER_VAL_LOCKS_EXT
#endif

// TODO: is not the addr but the index in the account array (TODO: subtract base addr)!

// Logs the read-set after acquiring the locks
// TODO: check write/write conflicts

#if CMP_TYPE == CMP_EXPLICIT
/* TODO: need logPos (do every thread commit the same number of transactions?) */
#define SET_ON_LOG(addr) \
	int *explicitLog = (int*)GPU_log->dev_rset; \
	unsigned logPos = explicitLogOffset + GPU_log->explicitLogOffThr[tid_]; \
	uintptr_t rsetAddr = (uintptr_t)(addr); \
	uintptr_t devBAddr = (uintptr_t)args->inBuf; \
	uintptr_t pos = (rsetAddr - devBAddr) >> PR_LOCK_GRAN_BITS; /* stores the index instead of the address */ \
	explicitLog[logPos + i] = pos+1 /* 0 is NULL */ \
//
/*if (GPU_log->explicitLogOffThr[tid_]==98) printf("[%i] explicitLogOffset=%i, explicitLogOffThr=%i, i=%i\n", (int)tid_,\
(int)explicitLogOffset, (int)GPU_log->explicitLogOffThr[tid_], i);*/ \
#elif CMP_TYPE == CMP_COMPRESSED
#define SET_ON_LOG(addr) \
	uintptr_t rsetAddr = (uintptr_t)(addr); \
	uintptr_t devBAddr = (uintptr_t)args->inBuf; \
	uintptr_t pos = (rsetAddr - devBAddr) >> PR_LOCK_GRAN_BITS; \
	unsigned short *RSetBitmap = (unsigned short*)GPU_log->dev_rset; \
	ByteM_SET_POS(pos, RSetBitmap) \
//
#else
// error or disabled
#define SET_ON_LOG(addr) /* empty */
#endif

#define PR_AFTER_VAL_LOCKS_EXT(args) ({ \
  int i; \
	HeTM_GPU_log_s *GPU_log = (HeTM_GPU_log_s*)args->pr_args_ext; \
	/* TODO: explicit log only */ \
	HeTM_GPU_log_explicit_before_reads \
	/* ---------------------- */ \
	for (i = 0; i < args->rset.size; i++) { \
		SET_ON_LOG(args->rset.addrs[i]); /* add read to devLogR */ \
	} \
	/* TODO: explicit logOnly */ \
	HeTM_GPU_log_explicit_after_reads /* offset of the next transaction */ \
	/* ---------------------- */ \
}) \

#ifdef PR_AFTER_WRITEBACK_EXT
#undef PR_AFTER_WRITEBACK_EXT
#endif

// Logs the write-set after acquiring the locks (TODO: it is the same in PR_AFTER_VAL_LOCKS_EXT)
#define PR_AFTER_WRITEBACK_EXT(args, i, addr, val) ({ \
	/* HeTM_GPU_log_s *GPU_log = (HeTM_GPU_log_s*)args->pr_args_ext; */ \
	/* SET_ON_LOG(addr); TODO: add write to BM */ \
}) \

#ifdef PR_AFTER_CPY_TO_HOST_EXT
#undef PR_AFTER_CPY_TO_HOST_EXT
#endif

#ifdef HETM_DEB
#define CPY_BACK_DEBUG() \
	memman_select("Stats_OnIntersect"); \
	memman_cpy_to_cpu(NULL); \
	memman_select("HeTM_dev_rset"); \
	memman_cpy_to_cpu(NULL) \
//
#else /* HETM_DEB */
#define CPY_BACK_DEBUG() /* empty */
#endif /* HETM_DEB */

#define PR_AFTER_CPY_TO_HOST_EXT(args) ({ \
	HeTM_GPU_log_explicit_teardown; \
	CPY_BACK_DEBUG(); \
	memman_select("HeTM_gpuLog"); \
	memman_free_dual(); \
}) \

#include "pr-stm-wrapper.cuh" // enables the granularity
#include "pr-stm-internal.cuh"

// --------------------
__constant__ __device__ unsigned PR_seed = 1234; // TODO: set seed

__global__ void setupKernel(void *args) {
	hiprandState *state = (hiprandState*)args;
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init(PR_seed, id, 0, &state[id]);
}

__device__ unsigned PR_i_rand(pr_tx_args_dev_host_s args, unsigned n)
{
	HeTM_GPU_log_s *GPU_log = (HeTM_GPU_log_s*)args.pr_args_ext;
	hiprandState *state = GPU_log->state;
	int id = PR_THREAD_IDX;
	int x = 0;
	hiprandState localState = state[id];
	x = hiprand(&localState);
	state[id] = localState;
	return x % n;
}
// --------------------

//versions in global memory	10 bits(version),0 bits (padding),20 bits(owner threadID),1 bit(LOCKED),1 bit(pre-locked)
#define	offVers	22
#define offOwn	2
#define offLock	1

#define getVersion(x)     ( ((x) >> offVers) )
#define checkPrelock(x)   ( (x) & 0b1)
#define checkLock(x)      ( ((x) >> offLock) & 0b1)
#define getOwner(x)       ( ((x) >> offOwn) & 0xfffff)
#define maskVersion       0xffc00000

#define finalIdx          (threadIdx.x+blockIdx.x*blockDim.x)
#define newLock(x,y,z)    ( ((x) << offVers) | ((y) << offOwn) | (z))

/****************************************************************************
 *	GLOBALS
 ****************************************************************************/

__constant__ __device__ long size;
__constant__ __device__ int  TransEachThread;
// __constant__ __device__ const int BANK_NB_TRANSFERS;
__constant__ __device__ int  hashNum;

/****************************************************************************
 *	KERNELS
 ****************************************************************************/

//   Memory access layout
// +------------+--------------------+
// | NOT ACCESS |      CPU_PART      |
// +------------+--------------------+
// +------------------+--------------+
// |     GPU_PART     |  NOT ACCESS  |
// +------------------+--------------+
//
// random Function random several different numbers and store them into idx(Local array which stores idx of every source in Global memory).
__device__ void random_Kernel(PR_txCallDefArgs, int *idx, hiprandState* state, int size)
{
	int i, j;
	HeTM_GPU_log_s *GPU_log = (HeTM_GPU_log_s*)args.pr_args_ext;
	int id = threadIdx.x+blockDim.x*blockIdx.x;

	// generates the target accounts for the transaction
	for (i = 0; i < BANK_NB_TRANSFERS; i++) {
		int m = 0;
		int is_intersect = IS_INTERSECT_HIT(PR_rand(100000));

		// accounts must be different
		while (m < 1) {
			int randVal = PR_rand(INT_MAX);
			// TODO: the size becomes useless
			if (is_intersect) {
				GPU_log->onIntersect[id]++;
				idx[i] = INTERSECT_ACCESS(randVal, size);
			} else {
				idx[i] = GPU_ACCESS(randVal, size);
			}
			bool hasEqual = 0;

			// idx array is traveled to check repeated accesses
			for (j = 0; j < i; j++)	{
				if (idx[i] == idx[j]) {
					hasEqual = 1;
					break;
				}
			}
			if (hasEqual != 1) {
				// if it is not repeated goto i++ in the outer for loop
				m++; // break while (m < 1)
			}
		}
	}
	/*idx[0] = generate_kernel(state,100)%size;
	for (int i = 0; i < BANK_NB_TRANSFERS; i++)
	{
	idx[i] = (idx[0]+i)%size;
	}*/
}


/*********************************
 *	bankTx()
 *
 *  Main PR-STM transaction kernel
 **********************************/
/*
* Isto e' o que interessa
*/
__global__ void bankTx(PR_globalKernelArgs)
{
	PR_enterKernel();

	int i = 0, j;	//how many transactions one thread need to commit
	int target;
	PR_GRANULE_T nval;
	int idx[BANK_NB_TRANSFERS];
	PR_GRANULE_T reads[BANK_NB_TRANSFERS];
	PR_GRANULE_T *accounts = (PR_GRANULE_T*)args.inBuf;
	size_t nbAccounts = args.inBuf_size / sizeof(PR_GRANULE_T);
	HeTM_GPU_log_s *GPU_log = (HeTM_GPU_log_s*)args.pr_args_ext;

	random_Kernel(PR_txCallArgs, idx, GPU_log->state, nbAccounts);	//get random index

  // TODO: it was TransEachThread * iterations
	while (i++ < TransEachThread) { // each thread need to commit x transactions
		PR_txBegin();

		// reads the accounts first, then mutates the locations
		for (j = 0; j < BANK_NB_TRANSFERS; j++)	{
			reads[j] = PR_read(accounts + idx[j]);
			if (pr_args.is_abort) break; // PR_txBegin is a simple while loop
		}

		if (pr_args.is_abort) { PR_txRestart(); } // optimization

		for (j = 0; j < BANK_NB_TRANSFERS / 2; j++) {
			target = j*2;
			nval = reads[target] - 1; // -money
			PR_write(accounts + idx[target], nval); //write changes to write set
			if (pr_args.is_abort) break;

			target = j*2+1;
			nval = reads[target] + 1; // +money
			PR_write(accounts + idx[target], nval); //write changes to write set
			if (pr_args.is_abort) break;
		}
		if (pr_args.is_abort) { PR_txRestart(); } // optimization
		PR_txCommit();
	}

	PR_exitKernel();
}


/*********************************
 *	readKernelTransaction()
 *
 *  Main PR-STM transaction kernel
 **********************************/
__global__
__launch_bounds__(1024, 1)
void HeTM_memcd_read(PR_globalKernelArgs)
  volatile long* a,	/*values in global memory*/
	int * ts_vec,										/*timestamp array*/
	cuda_output_t * output,								/*produced results*/
	volatile uint_64* mymutex,							/*store lock,version,owner in format version*10000+owner*10+lock*/
	int * dev_abortcount,								/*record how many times aborted*/
	unsigned int size,									/*size of the data-set*/
	int clock_value,									/*sync timestamp*/
	unsigned int * tx_queue,							/*transaction queue*/
	int * devLogR,
	int * devLogW)
{
  PR_enterKernel();

  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  memcd_input_s *input = (memcd_input_s*)args.inBuf;
  PR_GRANULE_T *accounts = input->accounts;
  int clock_value = input->clock_value;
  unsigned int *tx_queue = input->tx_queue;
  cuda_output_t *output  = (PR_GRANULE_T*)args.outBuf;

  // TODO: there was some optimization with the write-set
	int idx[NUMBER_WAYS]; // indices to read
	int value[NUMBER_WAYS]; // read values

	int goal = 0;

	rd_idx[0] = tx_queue[idx] * num_ways;
	for (int j = 1; j < num_ways; j++) {
    rd_idx[j] = rd_idx[0] + j;
  }

	goal = tx_queue[idx] & 0xffff;
	goal = (goal % num_ways);

	PR_txBegin();
		for (int j = 0; j <= goal; j++) {
      value[j] = PR_read(accounts + rd_idx[j]);
      if (pr_args.is_abort) break;
		}

		if (pr_args.is_abort) { PR_txRestart(); } // optimization
  PR_txCommit();

	output[finalIdx].key   = value[goal];
	output[finalIdx].index = idx[goal];
	ts_vec[rd_idx[goal]]   = clock_value;

  PR_exitKernel();
}

/*********************************
 *	writeKernelTransaction()
 *
 *  Main PR-STM transaction kernel
 **********************************/
__global__ void HeTM_memcd_write(PR_globalKernelArgs)

  volatile long* a,	/*values in global memory*/
	int * ts_vec,										/*timestamp array*/
	cuda_output_t * output,										/*produced results*/
	volatile uint_64* mymutex,							/*store lock,version,owner in format version*10000+owner*10+lock*/
	int * dev_abortcount,								/*record how many times aborted*/
	unsigned int size,									/*size of the data-set*/
	int clock_value,									/*sync timestamp*/
	unsigned int * tx_queue,							/*transaction queue*/
	int * devLogR,
	int * devLogW)
{
	// structure of read set and write set. Both of them have idx(d_idx),value(r_idx,w_idx) and version(rv_idx,wv_idx). And they are in local memory.
	int  rd_idx[NUMBER_WAYS];		//address in read set
	int  wt_idx[NUMBER_WAYS];		//address in write set

	long r_idx[NUMBER_WAYS];		//value in read set
	int  rv_idx[NUMBER_WAYS];		//version in read set
	long w_idx[NUMBER_WAYS];		//value in write set
	int  lv_idx[NUMBER_WAYS];		//version in lock set
	int  lc_idx[NUMBER_WAYS];		//lock set for address
	int  i = 0;						//how many transactions one thread need to commit

	int goal = 0;

	unsigned int logOffset = 0, trOffset = 0;

	rd_idx[0] = tx_queue[finalIdx] * num_ways;
	for (int j=1; j < num_ways; j++)
		rd_idx[j] = rd_idx[0] + j;


	goal =  tx_queue[finalIdx] & 0xffff;

	while (i < TransEachThread){	//each thread need to commit x transactions

		int rd_idx_size = 0;	//read set size
		int wt_idx_size = 0;	//write set size
		int lc_idx_size = 0;
		int min_val=-1, min_pos=0;

		bool flag = true;	//flag for whether succeessfully called one read or write or commit kernel
		bool flag2 = true;	//flag for whether succeessfully called all read or write or commit kernel

		/*Search hash table for an empty spot or an entry to evict*/
		for (int j=0; j < num_ways; j++) {
			flag = openRead_Kernel(a, mymutex, rd_idx, r_idx, rv_idx, rd_idx_size, 0); //read data and version from global memory to read set
			if (flag == false){	//if one read failed, increase abort time, change flag2 to false and exit this loop
				atomicAdd(&dev_abortcount[blockIdx.x], 1);	//record abort times
				//printf("Open Read Aborted,threadid = %d,\n",finalidx);
				flag2 = false;
				break;
			}
			//Check if it is free or if it is the same value
			if ( a[rd_idx[j]] == goal || ts_vec[rd_idx[j]] == 0 ) {
				min_pos = j;
				break;
			} else {
				if(  min_val == -1 || min_val > ts_vec[rd_idx[j]] ) {
					min_val = ts_vec[rd_idx[j]];
					min_pos = j;
				}
			}
		}
		wt_idx[0] = rd_idx[min_pos];
		r_idx[min_pos] = goal;

		if (flag2 == false)	// if one read failed, abort and try again
			continue;



		r_idx[min_pos] = goal;
		flag = openWrite_Kernel(mymutex, rd_idx, wt_idx, r_idx, w_idx, rv_idx, lv_idx, lc_idx, min_pos, rd_idx_size, wt_idx_size, lc_idx_size);	//write changes to write set
		if (flag == false){	//check if this open write succeed
			atomicAdd(&dev_abortcount[blockIdx.x], 1);	//record abort times
			//printf("Open Write Aborted,threadid = %d,\n",finalidx);
			flag2 = false;
			break;
		}

		if (flag2 == false)	// if one write failed, abort and try again
			continue;


		if (validate_Kernel(mymutex, rd_idx, rv_idx, lv_idx, lc_idx, rd_idx_size, lc_idx_size, devLogR, logOffset + trOffset) == false){	//validate all read data and lock write memory addresses
			atomicAdd(&dev_abortcount[blockIdx.x], 1);	//record abort times
			//printf("Validate Aborted,threadid = %d\n",finalidx);
			continue;
		}
		else	//if succeessfully validated, then commit
		{
			//printf("Validate succeed, threadIDX= %d, write_idx[0] = %d, write_idx[1] = %d, times = %d, a[0] = %d, a[1] = %d, a[2] = %d, wt[0] = %d, wt[1] = %d \n",finalIdx,wt_idx[0],wt_idx[1],*dev_abortcount,a[0],a[1],a[2],w_idx[0],w_idx[1]);
			//__threadfence();
			commit_Kernel(a, wt_idx, mymutex, w_idx, lv_idx, lc_idx, wt_idx_size, lc_idx_size, devLogW, devLogR, logOffset + trOffset);	//commit from write set to global memory
			output[finalIdx].key = w_idx[0];
			output[finalIdx].index = wt_idx[0];
			ts_vec[rd_idx[min_pos]] = clock_value;
			i++;

		}
	}
}

/****************************************************************************
 *	FUNCTIONS
/****************************************************************************/

extern "C"
cuda_config cuda_configInit(int size, int ways, int trans, int hash, int tx, int bl) {
	cuda_config c;

	c.size = size;
	c.num_ways = ways > 0 ? ways : NUMBER_WAYS;
	c.TransEachThread = trans > 0 ? trans : DEFAULT_TransEachThread;
	c.hashNum = hash > 0 ? hash : DEFAULT_hashNum;
	c.threadNum = tx > 0 ? tx : DEFAULT_threadNum;
	c.blockNum = bl > 0 ? bl : DEFAULT_blockNum;

	return c;
}

extern "C"
hipError_t cuda_configCpy(cuda_config c) {
	hipError_t cudaStatus;
	int err = 1;

	while (err) {
		err=0;
		void * point1 = &c.size;
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dataset_size), point1, sizeof(long),0, hipMemcpyHostToDevice );
		if ( cudaStatus  != hipSuccess) {
			printf("hipMemcpy to device failed for size!");
			continue;
		}
		void * point2 = &c.TransEachThread;
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(TransEachThread), point2, sizeof(int),0, hipMemcpyHostToDevice );
		if ( cudaStatus  != hipSuccess) {
			printf("hipMemcpy to device failed for TransEachThread!");
			continue;
		}
		void * point3 = &c.hashNum;
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(hashNum), point3, sizeof(int),0, hipMemcpyHostToDevice );
		if ( cudaStatus  != hipSuccess) {
			printf("hipMemcpy to device failed for hashNum!");
			continue;
		}
		void * point4 = &c.num_ways;
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(num_ways), point4, sizeof(int),0, hipMemcpyHostToDevice );
		if ( cudaStatus  != hipSuccess) {
			printf("hipMemcpy to device failed for num_ways!");
			continue;
		}
	}
	/*hipMemcpyFromSymbol(&c.hashNum,HIP_SYMBOL("hashNum"),sizeof(int));
	printf("hashNum: %d\n",c.hashNum);*/

	return cudaStatus;
};
