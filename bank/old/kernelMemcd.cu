#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime_api.h"
#include "helper_timer.h"
#include <time.h>

#include "memman.h"
#include "knlman.h"

#include "setupKernels.cuh"

extern "C" {
#include "cuda_wrapper.h"
}

#include "bankKernel.cuh"

//#define	DEBUG_CUDA
//#define	DEBUG_CUDA2

//Support for the lazylog implementation
// moved to cmp_kernels.cuh
// #define EXPLICIT_LOG_BLOCK     (TransEachThread * BANK_NB_TRANSFERS)
// #define EXPLICIT_LOG_SIZE      (blockNum * threadNum * EXPLICIT_LOG_BLOCK)	//size of the lazy lock

//Support for the compressed log implementation
#define readMask         0b01	//Set entry to write
#define writeMask        0b10	//Set entry to read

//versions in global memory	10 bits(version),0 bits (padding),20 bits(owner threadID),1 bit(LOCKED),1 bit(pre-locked)
#define	offVers	22
#define offOwn	2
#define offLock	1

#define finalIdx          (threadIdx.x+blockIdx.x*blockDim.x)
#define newLock(x,y,z)    ( ((x) << offVers) | ((y) << offOwn) | (z))

#define uint_64				int

#define CHECK_ERROR_CONTINUE(cuda_call) \
  cudaStatus = cuda_call; \
  if (cudaStatus != hipSuccess) { \
    printf("Error " #cuda_call " \n" __FILE__ ":%i\n   > %s\n", \
      __LINE__, hipGetErrorString(cudaStatus)); \
    /*goto Error;*/ \
    continue; \
  } \
//

void CUDART_CB checkCallback(hipStream_t stream, hipError_t status, void *data)
{
  stream_t *st = (stream_t*)data;
  int i;

  if(status != hipSuccess) {
    st->count = st->maxC;
    st->isCudaError = -1;
    printf("Comparison kernel crashed. Error code is: %s.\n", hipGetErrorString(status));
  }

  // clears the ad-hoc memory
  memman_ad_hoc_free((void*)stream);

  pthread_mutex_lock(&st->mutex);
  i = st->count;
  if(i>=st->maxC)
    printf("BUG: st->count = %d\n",i);
  st->count++;
  pthread_mutex_unlock(&st->mutex);
}


/* ################################################################################################# *
 * HOST CODE
 * ################################################################################################# */

/****************************************
 *	jobWithCuda_init(size,hostLogMax)
 *
 *	Description:	Initialize the GPU, by allocating all necessary memory,
 *					transferring static data and running the setup kernel.
 *
 *	Args:
 *		int size		: Size (in integers) to allocate for working set data
 *		long ** accounts: Pointer to host array pointer
 *      int hostLogMax	: Maximum number of entries the host transaction log can contain
 *		long * b:		:(Optional) Host log address, for use with zero copy
 *
 *	Returns:
 *		cuda_t: 	Custom structure containing all essential CUDA pointers/data
 *					or null in case of failure
 *
 ****************************************/
 // TODO: put GRANULE_T or account_t
extern "C"
cuda_t * jobWithCuda_init(account_t *accounts, int size, int trans, int hash, int tx, int bl)
{
  //int *a = (int *)malloc(size * sizeof(int));
  size_t accountsSize, cuRandSize, nbAccounts, nbGPUThreads;
  int *dev_bm = 0;         //compressed log array
  int *dev_LogR = 0, *dev_LogW = 0;	//Log array
  cuda_config cuda_info;    //Cuda config info
  bool err = 1;	// Replacement for the goto (TODO: why loop on error? Just exit!)
  cuda_t *c_data;
  hipError_t cudaStatus;
  hiprandState *devStates;	//randseed array

  //hipProfilerStop();							//Stop unnecessary  profiling

  cuda_info = cuda_configInit(size, trans, hash, tx, bl);

  PR_init(); // inits PR-STM mutex array

  nbAccounts   = cuda_info.size;
  nbGPUThreads = cuda_info.blockNum * cuda_info.threadNum;
  cuRandSize   = nbGPUThreads * sizeof(hiprandState);
  accountsSize = nbAccounts * sizeof(account_t);

  queue_t * q = NULL;

  // Init check Tx kernel
  // TODO: init EXPLICIT
  HeTM_setup_checkTxCompressed();
  HeTM_setup_checkTxExplicit();
  HeTM_setup_finalTxLog2();
  HeTM_setup_bankTx();

  while (err) {
    err = false;

    // Choose which GPU to run on, change this on a multi-GPU system.
    CHECK_ERROR_CONTINUE(hipSetDevice(DEVICE_ID));

    CHECK_ERROR_CONTINUE(hipMalloc((void **)&devStates, cuRandSize));
    setupKernel <<< cuda_info.blockNum, cuda_info.threadNum >>>(devStates); /* setups PR_rand on GPU */
  	hipDeviceSynchronize();

    memman_alloc_gpu("HeTM_accounts_a", accountsSize, accounts, 0);
    memman_zero_gpu(NULL); // copy to GPU
    memman_alloc_gpu("HeTM_accounts_b", accountsSize, accounts, 0);
    memman_zero_gpu(NULL);
    memman_alloc_gpu("HeTM_accounts_bckp", accountsSize, accounts, 0);


    // TODO
    memman_alloc_gpu("HeTM_accounts_ts", accountsSize, accounts, 0);
    memman_zero_gpu(NULL);

    memman_alloc_dual("HeTM_cpu_versions", nbAccounts * sizeof(long), 0);
    memman_zero_gpu(NULL);

    // TODO: use alloc_gpu
    // memman_alloc_dual("HeTM_CPU_wset_log", LOG_SIZE * sizeof(HeTM_CPULogEntry), 0);
    // CHECK_ERROR_CONTINUE(hipMalloc((void **)&stm_log, LOG_SIZE * sizeof(HeTM_CPULogEntry)));

    //Comparison flag allocation
    memman_alloc_dual("HeTM_flag_inter_conflict", sizeof(int), 0);

    CHECK_ERROR_CONTINUE(hipMalloc((void **)&dev_LogR, nbAccounts * sizeof(int)));
    CHECK_ERROR_CONTINUE(hipMemset(dev_LogR, 0, nbAccounts * sizeof(int)));

    CHECK_ERROR_CONTINUE(hipMalloc((void **)&dev_bm, nbAccounts * sizeof(long)));

    // 1 bit per accounts, i.e., allocates ceilling(nbAccounts/ bitsInByte)
#if HETM_CMP_TYPE == HETM_CMP_EXPLICIT
    memman_alloc_dual("HeTM_dev_rset", EXPLICIT_LOG_SIZE(cuda_info.blockNum, cuda_info.threadNum)*sizeof(int), 0);
    memman_zero_gpu(NULL);
#elif HETM_CMP_TYPE == HETM_CMP_COMPRESSED
    memman_alloc_dual("HeTM_dev_rset", nbAccounts / 8 + 1, 0);
    memman_zero_gpu(NULL);
#else
    // error or disabled
#endif

    CHECK_ERROR_CONTINUE(cuda_configCpy(cuda_info));

    time_t t;
    time(&t);
  }

  if (cudaStatus != hipSuccess) {
    printf("\nSetup: Error code is: %s\n", hipGetErrorString(cudaStatus));
    memman_select("HeTM_flag_inter_conflict");
    memman_free_dual();
    hipFree(devStates);
    // memman_select("HeTM_CPU_wset_log");
    // memman_free_dual(); // TODO: use GPU only
    // hipFree(stm_log);
    hipFree(dev_LogR);
    hipFree(dev_LogW);
    hipFree(dev_bm);
    memman_select("Stats_OnIntersect");
    memman_free_dual();
    memman_select("HeTM_dev_rset");
    memman_free_dual();
    memman_select("HeTM_cpu_versions");
    memman_free_dual();
    memman_select("HeTM_accounts_a");
    memman_free_gpu();
    memman_select("HeTM_accounts_b");
    memman_free_gpu();
    PR_teardown();
    c_data = NULL;
  } else {
    //Save cuda pointers
    c_data = (cuda_t *)malloc( sizeof(cuda_t) );

    c_data->host_a = accounts;
    memman_select("HeTM_accounts_a");
    c_data->dev_a = (account_t*)memman_get_gpu(NULL);
    memman_select("HeTM_accounts_b");
    c_data->dev_b = (account_t*)memman_get_gpu(NULL);
    memman_select("HeTM_accounts_bckp");
    c_data->dev_bckp = (account_t*)memman_get_gpu(NULL);
    c_data->dev_LogR = dev_LogR;
    c_data->dev_LogW = dev_LogW;
    c_data->devStates = (void*)devStates;
    c_data->size = cuda_info.size;
    c_data->dev_bm = dev_bm; // TODO
    // memman_select("HeTM_CPU_wset_log");
    // c_data->host_log = (HeTM_CPULogEntry*)memman_get_gpu(NULL);
    memman_select("HeTM_flag_inter_conflict");
    c_data->dev_flag = (int*)memman_get_gpu(NULL);
    c_data->dev_zc = NULL;
    c_data->bm_size = (cuda_info.size>>BM_HASH)+1;
    c_data->threadNum = cuda_info.threadNum;
    c_data->blockNum = cuda_info.blockNum;
    c_data->TransEachThread=cuda_info.TransEachThread;
  }

  return c_data;
}

/****************************************
 *	jobWithCuda_initStream()
 *
 *	Description:
 *
 *
 *	Args:

 *
 *	Returns:
 *		stream_t:
 *
 *
 ****************************************/
extern "C"
stream_t *jobWithCuda_initStream(cuda_t *d, int id, int count) {
  stream_t *stream_data = NULL;
  // hipError_t cudaStatus;
  // HeTM_CPULogEntry * host_point = 0, * stream_point = 0;
  // size_t logSize = LOG_SIZE * sizeof(HeTM_CPULogEntry);
  int n = 0;

  bool err = 1;	//Replacement for the goto

  // per thread
  stream_data = (stream_t*)malloc( sizeof(stream_t) );

  // Each CPU thread allocates some space in the GPU to dump
  // its thread local STM log.
  // memman_alloc_gpu("HeTM_CPU_wset_log", logSize, NULL, MEMMAN_THRLC);
  // stream_data->host_log = (HeTM_CPULogEntry*)memman_get_gpu(NULL);

  //Save pointers
  stream_data->st = NULL; //(hipStream_t *)malloc( sizeof(hipStream_t)*count );
  stream_data->isCudaError = 0;
  stream_data->id = id;
  stream_data->maxC = count;
  stream_data->count = 0;
  pthread_mutex_init(&stream_data->mutex, NULL);

  for (n = 0; n < count; n++) {
    // hipStreamCreate(&stream_data->st[n]);
    knlman_add_stream();
  }

  return stream_data;
}

/****************************************
 *	jobWithCuda_run(d,a)
 *
 *	Description:	Update working set data and run transaction kernel.
 *					Failures are only detected on subsequent calls to jobWithCuda_wait()
 *
 *	Args:
 *		cuda_t * d		: Custom structure containing all essential transaction kernel CUDA pointers/data
 *      long * a		: Working set data
 *
 *	Returns:
 *		int:		1 in case of success, 0 otherwise
 *
 ****************************************/
extern "C"
int jobWithCuda_run(cuda_t *d, account_t *a) {
  bool err = 1;
  hipError_t cudaStatus;

  while (err) {
    err = 0;

    CHECK_ERROR_CONTINUE(hipSetDevice(DEVICE_ID));

    knlman_select("HeTM_bankTx");
    knlman_set_nb_blocks(d->blockNum, 1, 1);
    knlman_set_nb_threads(d->threadNum, 1, 1);

    HeTM_bankTx_s bankTx_args = {
      .knlArgs = {
        .d = d,
        .a = a,
      },
      .clbkArgs = NULL
    };
    knlman_set_entry_object(&bankTx_args);
    knlman_run();

    //Check for errors
    cudaStatus = hipGetLastError();
  }

  if (cudaStatus != hipSuccess) {
    printf("\nTransaction kernel launch failed. Error code: %s.\n", hipGetErrorString(cudaStatus));
    return 0;
  }

  return 1;
}

/****************************************
 *	jobWithCuda_wait()
 *
 *	Description:	Wait for cuda execution to conclude.
 *
 *	Args: 			(none)
 *
 *	Returns:		(none)
 *
 ****************************************/
extern "C"
void jobWithCuda_wait(){

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  hipError_t cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    printf("hipDeviceSynchronize returned error code: %d\n.", cudaStatus);
  }
}

/****************************************
 *	jobWithCuda_checkStream(d,vec,size_stm,id,streamtime,trf)
 *
 *	Description:	Copy host log data to device, configure and launch comparison kernel
 *					to detect conflicts between host and device transactions using the
 *					designated stream.
 *
 *	Args:
 *		cuda_t d	: Custom structure containing all essential CUDA pointers/data
 *		HeTM_CPULogEntry * vec	: Array containing host log
 *		int size_stm: Number of valid entries in the host log array
 *		float * time: (Optional) Pointer to store duration of comparison kernel
 *		float * trf : (Optional) Pointer to store duration of comparison kernel
 *		int id		: Id of the launching thread
 *		hipStream_t stream: Cuda stream to launch to
 *
 *	Returns:
 *		int:		Result of the comparison: 0 if comparison detected no conflicts,
 *					1 otherwise or -1 in case of error.
 *
 ****************************************/
extern "C"
int jobWithCuda_checkStream(cuda_t d, stream_t *st, HeTM_CPULogEntry *vec, int size_stm, int n) {
  hipError_t cudaStatus;
  bool err = 1;
  int ret = 1; //Return value
  HeTM_CPULogEntry * streamHLog; // on device
  hipStream_t stream;
  size_t logSize = size_stm * sizeof(HeTM_CPULogEntry);

  if (size_stm==0) {
    __sync_add_and_fetch(&st->count, 1);
    return 0;
  }
  //printf("Comparison size: %d\n", size_stm);
  while (err) {
    err=0;

    knlman_choose_stream(n);
    // knlman_choose_next_stream();
    // knlman_sync_stream(); // synchronize threads
    stream = (hipStream_t)knlman_get_current_stream();

    streamHLog = (HeTM_CPULogEntry*)memman_ad_hoc_alloc((void*)stream, (void*)vec, logSize);
    memman_ad_hoc_cpy((void*)stream);

    memman_select("HeTM_dev_rset");
    void *rset = memman_get_gpu(NULL);
    memman_select("HeTM_cpu_versions");
    long *vers  = (long*)memman_get_gpu(NULL);

#if HETM_CMP_TYPE == HETM_CMP_COMPRESSED
    // -----------------------------------------------
    //Calc number of blocks
    int bo = (size_stm + 31) / (32);
    dim3 blocksCheck(bo); // partition the stm_log by the different blocks
    dim3 threadsPerBlock(32); // each block has 32 threads

    knlman_select("HeTM_checkTxCompressed");
    knlman_set_nb_blocks(bo, 1, 1);
    knlman_set_nb_threads(32, 1, 1);

    // Memory region of the entry object
    HeTM_checkTxCompressed_s checkTxCompressed_args = {
      .knlArgs = {
        .dev_flag = d.dev_flag,
        .stm_log  = streamHLog,
        .size_stm = size_stm,
        .size_log = d.size,
        .mutex    = PR_lockTableDev,
        .devLogR  = (int*)rset,
        .a        = d.dev_a,
        .b        = d.dev_b,
        .vers     = vers,
      },
      .clbkArgs = st
    };
    knlman_set_entry_object(&checkTxCompressed_args);

    knlman_run();
    // -----------------------------------------------
#elif HETM_CMP_TYPE == HETM_CMP_EXPLICIT
    int xThrs = CMP_EXPLICIT_THRS_PER_RSET / CMP_EXPLICIT_THRS_PER_WSET;
    int yThrs = CMP_EXPLICIT_THRS_PER_WSET;

    int hasRemainderXThrs = EXPLICIT_LOG_SIZE(d.blockNum, d.threadNum) % xThrs;

    int xBlocks = EXPLICIT_LOG_SIZE(d.blockNum, d.threadNum) / xThrs;
    int yBlocks = size_stm / CMP_EXPLICIT_THRS_PER_WSET;

    int hasRemainderYBlocks = size_stm % CMP_EXPLICIT_THRS_PER_WSET;

    if (hasRemainderXThrs) xBlocks++;
    if (hasRemainderYBlocks) yBlocks++;

    knlman_select("HeTM_checkTxExplicit");
    knlman_set_nb_blocks(xBlocks, yBlocks, 1);
    knlman_set_nb_threads(xThrs, yThrs, 1);

    HeTM_checkTxExplicit_s checkTxExplicit_args = {
      .knlArgs = {
        .dev_flag = d.dev_flag,
        .stm_log  = streamHLog,
        .size_stm = size_stm,
        .size_logR= EXPLICIT_LOG_SIZE(d.blockNum, d.threadNum),
        .devLogR  = (int*)rset,
        .mutex    = PR_lockTableDev,
        .a        = d.dev_a,
        .b        = d.dev_b,
        .vers     = vers,
      },
      .clbkArgs = st
    };
    knlman_set_entry_object(&checkTxExplicit_args);

    knlman_run();
#else
  printf("Error! no compare method selected!\n");
#endif

    CHECK_ERROR_CONTINUE(hipGetLastError());
  }

  // Check for errors
  cudaStatus = hipGetLastError(); //synchronize threads
  if (cudaStatus != hipSuccess) {
    printf("Stream CMP[%d]>> Error code is: %s.\n", st->id, hipGetErrorString(cudaStatus));
    ret = -1;
  }

  return ret;
}


/****************************************
 *	jobWithCuda_swap(d,a)
 *
 *	Description:	Overwrites devices working set with the hosts
 *
 *	Args:
 *		cuda_t  d		: Custom structure containing all essential transaction kernel CUDA pointers/data
 *
 *	Returns:
 *		long *:			0 in case of failure, a pointer otherwise
 *
 ****************************************/
extern "C"
account_t* jobWithCuda_swap(cuda_t d){
  return d.host_a;
}

/****************************************
 *	jobWithCuda_dupd(d,b)
 *
 *	Description:	Transfers host values to device.
 *
 *	Args:
 *		cuda_t d		: Custom structure containing all essential transaction kernel CUDA pointers/data
 *      long * b		: Host working set data
 *
 *	Returns:
 *		int:		1 in case of success, 0 otherwise
 *
 ****************************************/
extern "C"
int jobWithCuda_dupd(cuda_t d, account_t *b)
{
  hipError_t cudaStatus;

  // Transfer comparison flag to device
  memman_select("HeTM_flag_inter_conflict");
  memman_zero_gpu(NULL); //copy new vector from CPU to GPU
  // cudaStatus = hipMemset(d.dev_flag, 0, sizeof(int));
  // if (cudaStatus != hipSuccess) {
  //   printf("hipMemcpy to device failed for dev_flag!\n");
  //   return 0;
  // }

  // Transfer comparison flag to device
  // cudaStatus = hipMemset(PR_lockTableDev, 0, PR_LOCK_TABLE_SIZE*sizeof(int));	//copy new vector from CPU to GPU
  if (cudaStatus != hipSuccess) {
    printf("hipMemcpy to device failed for PR_lockTableDev!\n");
    return 0;
  }

  return 1;
}

/****************************************
 *	jobWithCuda_hupd(d,vec,size_stm,time,ab.com)
 *
 *	Description:	Copy results produced by device to host
 *
 *	Args:
 *		cuda_t * d	: Custom structure containing all essential CUDA pointers/data
 *		long * a	: Device updated working set
 *		int * bm	: Bitmap indicating updated memory positions
 *
 *	Returns:
 *		int:		Result of the comparison: 0 if comparison detected no conflicts,
 *					1 otherwise or in case of error.
 *
 ****************************************/
extern "C"
int jobWithCuda_hupd(cuda_t *d, account_t *a, int *bm)
{
  // TODO: check what bm is doing
  hipError_t cudaStatus;
  int err = 1;

  while(err) {
    err = 0;

    CHECK_ERROR_CONTINUE(hipDeviceSynchronize());

    // TODO: manter este
    //Transfer bitmap
    if (bm == NULL) {
      //Transfer data
      CHECK_ERROR_CONTINUE(hipMemcpy(a, d->dev_a, d->size*sizeof(long), hipMemcpyDeviceToHost)); //copy results from GPU to CPU
    } else {
      int i,j;
      for(i = 0; i< d->bm_size-1; i++) {
        if(bm[i]==1) {
          j=i<<BM_HASH;
          cudaStatus = hipMemcpyAsync(&a[j], &d->dev_a[j], BM_HASH_SIZE*sizeof(long), hipMemcpyDeviceToHost, 0);	//copy compressed log from GPU to CPU
          if (cudaStatus != hipSuccess) {
            printf("hipMemcpy failed for bm!");
            break;
          }
        }
      }
      j=i<<BM_HASH;
      if(bm[i]==1){
        CHECK_ERROR_CONTINUE(hipMemcpyAsync(&a[j], &d->dev_a[j], (d->size - j)*sizeof(long), hipMemcpyDeviceToHost, 0));	//copy compressed log from GPU to CPU
      }
    }
  }

  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    printf("\nCpyBack: Error code is: %s.\n", hipGetErrorString(cudaStatus));
    //printf("\nError code is: %s\n", hipGetErrorString(cudaStatus));
    return 0;
  }

  //These will be swaped again on kernel launch, need to fix this (TODO
  account_t *buff = d->dev_a;
  d->dev_a = d->dev_b;
  d->dev_b = buff;

  return 1;
}

/****************************************
 *	jobWithCuda_getStats(cd,ab,com)
 *
 *	Description:	Get cuda stats
 *
 *	Args:
 *		cuda_t * d	: Custom structure containing all essential CUDA pointers/data
 *		int * ab	: (Optional) Pointer to store tx kernel abort counter
 *		int * com	: (Optional) Pointer to store tx kernel commit counter
 *
 *	Returns:
 *		(None)
 *
 ****************************************/
extern "C"
void jobWithCuda_getStats(cuda_t d, long *ab, long *com) {
  hipError_t cudaStatus;
  int err = 1;

  while(err) {
    err = 0;

    CHECK_ERROR_CONTINUE(hipDeviceSynchronize());

    //Transfer aborts
    // TODO: WTF is this !!!!
    if (ab != NULL) {
      *ab = PR_nbAborts;
    }

    //Transfer commits
    if (com != NULL) {
      *com = PR_nbCommits;
    }
  }

  if (cudaStatus != hipSuccess) {
    printf("\nStats: Error code is: %s.\n", hipGetErrorString(cudaStatus));
    return;
  }
}

/****************************************
 *	jobWithCuda_exit(d)
 *
 *	Description:	Finish Cuda execution, free device memory and reset device.
 *
 *	Args:
 *		cuda_t d	: Custom structure containing all essential CUDA pointers/data
 *
 *	Returns:		(none)
 *
 ****************************************/
extern "C"
void jobWithCuda_exit(cuda_t * d)
{
  hipError_t cudaStatus;

  cudaStatus = hipSetDevice(DEVICE_ID);
  if (cudaStatus != hipSuccess) {
    printf("hipDeviceSynchronize returned error code: %d\n", cudaStatus);
  }

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    printf("hipDeviceSynchronize returned error code: %d\n", cudaStatus);
  }

  if(d != NULL) {
    memman_select("HeTM_dev_rset");
    memman_free_dual();
    memman_select("HeTM_accounts_a");
    memman_free_gpu();
    memman_select("HeTM_accounts_b");
    memman_free_gpu();
    memman_select("HeTM_accounts_bckp");
    memman_free_gpu();
    // TODO:
    // memman_select("HeTM_CPU_wset_log");
    // memman_free_gpu();
    memman_select("HeTM_flag_inter_conflict");
    memman_free_dual();

    // hipFree(d->dev_a);
    // hipFree(d->dev_b);
    hipFree(d->dev_bm);
    hipFree(d->devStates);
    // hipFree(d->host_log);
    // hipFree(d->dev_flag);
    hipFree(d->dev_LogW);
    hipFree(d->dev_LogR);
    PR_teardown();
  }

  HeTM_teardown_checkTxCompressed();
  HeTM_teardown_bankTx();
  HeTM_teardown_finalTxLog2();
  HeTM_teardown_checkTxExplicit();

  // hipDeviceReset(); // This is crashing on CUDA 9.0

  return;
}

/****************************************
 *	CudaMallocWrapper(s,type)
 *
 *	Description:	CudaHostAlloc wrapper function
 *
 *	Args:
 *		size_t s	: Size of the memory to allocate
 *
 *	Returns:		Pointer to allocd memory
 *
 ****************************************/
extern "C"
long * CudaMallocWrapper(size_t s) {
  long * ret = 0;

  hipError_t cudaStatus = hipHostAlloc((long **)&ret, s, hipHostMallocPortable);
  if (cudaStatus != hipSuccess) {
    printf("hipHostAlloc returned error code: %d.\n", cudaStatus);
    return 0;
  }
/*#else
  ret = (long *)malloc(s);*/

  return ret;
}

/****************************************
 *	CudaZeroCopyWrapper(p)
 *
 *	Description:	hipHostGetDevicePointer wrapper function
 *
 *	Args:
 *		void *p		: Pointer to memory to be transfered
 *
 *	Returns:		Pointer to device allocd memory
 *
 ****************************************/
extern "C"
long * CudaZeroCopyWrapper(long * p) {
  long * ret = 0;

  ret=p;
  return ret;
}

/****************************************
 *	CudaFreeWrapper(*p)
 *
 *	Description:	CudaFreeAlloc wrapper function
 *
 *	Args:
 *		void *p		: Pointer to memory to be free'd
 *
 *	Returns:		(Nothing)
 *
 ****************************************/
extern "C"
void CudaFreeWrapper(void * p) {

//#if ZERO_CPY==1
  hipHostFree(p);

  return;
}

extern "C"
int jobWithCuda_bm(cuda_t d, int *bm)
{
  hipError_t cudaStatus;
  int err=1;

  while(err) {
    err = 0;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
      printf("jobWithCuda_bm() detected previous error!");
      //goto Error;
      continue;
    }

    knlman_select("HeTM_finalTxLog2");

    /* Define Kernel Size */
    knlman_set_nb_blocks((d.size+1023)/(1024), 1, 1);
    knlman_set_nb_threads(1024, 1, 1);

    HeTM_knl_finalTxLog2_s finalTxLog2_args = {
      .global_bitmap = d.dev_bm,
      .size          = d.size,
      .devLogR       = d.dev_LogR
    };
    knlman_set_entry_object(&finalTxLog2_args);
    knlman_run();
    hipDeviceSynchronize();

    //Transfer result
    cudaStatus = hipMemcpy(bm, d.dev_bm, d.bm_size*sizeof(int), hipMemcpyDeviceToHost);	//copy abortcounts
    if (cudaStatus != hipSuccess) {
      printf("hipMemcpy failed for bm!");
      continue;
    }

  }

  if (cudaStatus != hipSuccess) {
    printf("\nBM: Error code is: %s.\n", hipGetErrorString(cudaStatus));
    //printf("\nError code is: %s\n", hipGetErrorString(cudaStatus));
    return 0;
  }
  return 1;
}

extern "C"
int jobWithCuda_checkStreamFinal(cuda_t d, stream_t *st, int n)
{
  hipError_t cudaStatus;
  int *isConflict;

  st->count = 0;
  __sync_synchronize();

  memman_select("HeTM_flag_inter_conflict");
  memman_cpy_to_cpu(NULL); /* returns whether the comparison was successful */
  isConflict = (int*)memman_get_cpu(NULL);

  //Synchronize
  cudaStatus = hipStreamSynchronize(0);
  if (cudaStatus != hipSuccess) {
    printf("\nFinal Stream: Error code is: %s.\n", hipGetErrorString(cudaStatus));
    return 0;
  }

  return *isConflict;
}

extern "C"
void jobWithCuda_exitStream(stream_t * s)
{
  // Destroy streams
  knlman_destroy_streams();

  knlman_destroy_thread();

  // Free everything else
  // memman_select("HeTM_CPU_wset_log");
  // memman_free_gpu();

  pthread_mutex_destroy(&s->mutex);
  free(s);

  return;
}

/****************************************
 *	jobWithCuda_backup(d)
 *
 *	Description:	Backup GPU produced results
 *
 *	Args:
 *		cuda_t d	: Custom structure containing all essential CUDA pointers/data
 *
 *	Returns:		(none)
 *
 ****************************************/
extern "C"
void jobWithCuda_backup(cuda_t * d) {

  hipError_t cudaStatus = hipMemcpy(d->dev_bckp, d->dev_a, d->size * sizeof(account_t), hipMemcpyDeviceToDevice);
  if (cudaStatus != hipSuccess) {
    printf("Backup: hipMemcpy failed for dev_bckp. Error code %d: %s.\n", cudaStatus, hipGetErrorString(cudaStatus));
  }
}

/****************************************
 *	jobWithCuda_backupRestore(d)
 *
 *	Description:	Restore GPU backup of produced results
 *
 *	Args:
 *		cuda_t d	: Custom structure containing all essential CUDA pointers/data
 *
 *	Returns:		(none)
 *
 ****************************************/
extern "C"
void jobWithCuda_backupRestore(cuda_t * d) {

  hipError_t cudaStatus = hipMemcpy(d->dev_a, d->dev_bckp, d->size * sizeof(account_t), hipMemcpyDeviceToDevice);
  if (cudaStatus != hipSuccess) {
    printf("BRestore: hipMemcpy failed for dev_a. Error code %d: %s.\n", cudaStatus, hipGetErrorString(cudaStatus));
  }
}
