#include "hip/hip_runtime.h"
#include "hetm-log.h"
#include "hetm.cuh"
#include "stm-wrapper.h"
#include "stm.h" // depends on STM
#include "knlman.h"
#include "hetm-cmp-kernels.cuh"
// #include ".h" // depends on STM

#include <list>
#include <mutex>

#define NVTX_PROF_BLOCK   0
#define NVTX_PROF_BACKOFF 1

static std::list<HeTM_callback> beforeCPU, afterCPU;
std::mutex HeTM_statsMutex; // extern in hetm-threading-gpu

#define MAXIMUM_THREADS 1024

// TODO: still too many memcpyies

static void initThread(int id, void *data);
static void exitThread(int id, void *data);

#if HETM_LOG_TYPE != HETM_BMAP_LOG
static int consecutiveFlagCpy = 0; // avoid consecutive copies

thread_local static int inBackoff = 0;
thread_local static int nbCpyRounds = 0;
thread_local static int doneWithLog = 0;

static int launchCmpKernel(HeTM_thread_s*, size_t wsetSize);
static void checkCmpDone();
static void cmpBlockApply();
static void cpyWSetToGPU();
static void asyncCpy(void *argsPtr);
static void asyncCmp(void *argsPtr);
static void asyncGetInterConflFlag(void*);
#endif /* HETM_LOG_TYPE != HETM_BMAP_LOG */

void HeTM_cpu_thread()
{
  int threadId = HeTM_thread_data->id;
  HeTM_callback callback = HeTM_thread_data->callback;
  void *clbkArgs = HeTM_thread_data->args;

  // TODO: check order
  TM_INIT_THREAD(HeTM_shared_data.hostMemPool, HeTM_shared_data.sizeMemPool);
  initThread(threadId, clbkArgs);

  HETM_DEB_THRD_CPU("starting CPU worker %i", threadId);
  // printf("starting CPU worker %i\n", threadId);
  HeTM_sync_barrier();

  if (HeTM_shared_data.isGPUEnabled == 0) {
    // GPU threads is resposible for update statistics, but is off
    while (CONTINUE_COND) {
      callback(threadId, clbkArgs); // does 1 transaction
      HeTM_thread_data->curNbTxs++;
    }
    __sync_add_and_fetch(&HeTM_stats_data.nbTxsCPU, HeTM_thread_data->curNbTxs);
    __sync_add_and_fetch(&HeTM_stats_data.nbCommittedTxsCPU, HeTM_thread_data->curNbTxs);
    // HeTM_stats_data.nbDroppedTxsCPU == 0;
  } else {
    while (CONTINUE_COND) {
#if HETM_LOG_TYPE != HETM_BMAP_LOG
  #ifdef DISABLE_NON_BLOCKING
      cmpBlockApply(); // Block immediately
  #else /* VERS_DISABLE_NON_BLOCKING */
      checkCmpDone(); // Tests if ready
      cpyWSetToGPU();
  #endif /* VERS_DISABLE_NON_BLOCKING */
#else /* HETM_LOG_TYPE == HETM_BMAP_LOG */
      if (HeTM_get_GPU_status() == HETM_BATCH_DONE) {
        NVTX_PUSH_RANGE("blocked", NVTX_PROF_BLOCK);
        __sync_add_and_fetch(&HeTM_shared_data.threadsWaitingSync, 1);
        HeTM_sync_barrier(); // just block and let the GPU do its thing
        HeTM_sync_barrier();
        __sync_add_and_fetch(&HeTM_shared_data.threadsWaitingSync, -1);
        NVTX_POP_RANGE();
      }
#endif /* HETM_LOG_TYPE != HETM_BMAP_LOG */
      callback(threadId, clbkArgs); // does 1 transaction
      HeTM_thread_data->curNbTxs++;
      if (HeTM_get_GPU_status() == HETM_BATCH_DONE) {
        // transaction done while comparing
        HeTM_thread_data->curNbTxsNonBlocking++;
      }
    }
    NVTX_POP_RANGE();
  }

  HETM_DEB_THRD_CPU("exiting CPU worker %i", threadId);

  exitThread(threadId, clbkArgs);
  TM_EXIT_THREAD();
}

int HeTM_before_cpu_start(HeTM_callback req)
{
  beforeCPU.push_back(req);
  return 0;
}

int HeTM_after_cpu_finish(HeTM_callback req)
{
  afterCPU.push_back(req);
  return 0;
}

static void initThread(int id, void *data)
{
  knlman_add_stream(); // each thread has its stream
  HeTM_thread_data->stream = knlman_get_current_stream();
  stm_log_init();
  HeTM_thread_data->wSetLog = stm_thread_local_log;
  hipEventCreate(&HeTM_thread_data->cmpStartEvent);
  hipEventCreate(&HeTM_thread_data->cmpStopEvent);
  hipEventCreate(&HeTM_thread_data->cpyWSetStartEvent);
  hipEventCreate(&HeTM_thread_data->cpyWSetStopEvent);
  hipEventCreate(&HeTM_thread_data->cpyDatasetStartEvent);
  hipEventCreate(&HeTM_thread_data->cpyDatasetStopEvent);

  hetm_batchCount = &HeTM_shared_data.batchCount; // TODO: same code in GPU!!!

  for (auto it = beforeCPU.begin(); it != beforeCPU.end(); ++it) {
    HeTM_callback clbk = *it;
    clbk(id, data);
  }
}

static void exitThread(int id, void *data)
{
  HETM_DEB_THRD_CPU("Time cpy WSet = %10fms - Time cmp = %10fms \n"
    "Total empty space first chunk = %zu B\n",
    HeTM_thread_data->timeCpySum, HeTM_thread_data->timeCmpSum,
    HeTM_thread_data->emptySpaceFirstChunk);

  HeTM_statsMutex.lock();
  HeTM_stats_data.totalTimeCpyWSet += HeTM_thread_data->timeCpySum;
  HeTM_stats_data.totalTimeCmp += HeTM_thread_data->timeCmpSum;
  HeTM_stats_data.totalTimeCpyDataset += HeTM_thread_data->timeCpyDatasetSum;
  HeTM_stats_data.timeNonBlocking += HeTM_thread_data->timeBackoff;
  HeTM_stats_data.timeBlocking += HeTM_thread_data->timeBlocked;
  HeTM_statsMutex.unlock();

  for (auto it = afterCPU.begin(); it != afterCPU.end(); ++it) {
    HeTM_callback clbk = *it;
    clbk(id, data);
  }
}

#if HETM_LOG_TYPE != HETM_BMAP_LOG
static void asyncCpy(void *argsPtr)
{
  HeTM_thread_s *threadData = (HeTM_thread_s*)argsPtr;
#if HETM_LOG_TYPE == HETM_VERS2_LOG
  // TODO: 64 == upper bound of threads
  chunked_log_s truncated;

  threadData->curWSetSize = STM_LOG_BUFFER_SIZE*LOG_SIZE*LOG_ACTUAL_SIZE*sizeof(HeTM_CPULogEntry);
  truncated = MOD_CHUNKED_LOG_TRUNCATE(threadData->wSetLog, STM_LOG_BUFFER_SIZE);

  // size_t logSize=0;
  // chunked_log_node_s *node = threadData->wSetLog->buckets;
  // while (node != NULL) {
  //   logSize++;
  //   node = node->next;
  // }
  // printf("log size = %zu\n", logSize);

  if (truncated.first != NULL) {
    HeTM_wset_log_cpy_to_gpu(threadData, truncated.first, &threadData->curWSetSize);
    CHUNKED_LOG_DESTROY(&truncated);
  }
  consecutiveFlagCpy = 0; // allow to cpy the flag
  threadData->wSetLog->curr = truncated.first; // this one is to erase
#else /* HETM_LOG_TYPE == HETM_VERS_LOG */
  chunked_log_node_s *truncated;
  // truncates the exact amount
  truncated = stm_log_truncate(threadData->wSetLog);
  consecutiveFlagCpy = 0; // allow to cpy the flag
  HeTM_wset_log_cpy_to_gpu(threadData, truncated, &threadData->curWSetSize);
  threadData->wSetLog->curr = truncated; // this one is to erase
#endif /* HETM_LOG_TYPE == HETM_VERS_LOG */
  HETM_DEB_THRD_CPU("Buffered WSet of size %zu\n", threadData->curWSetSize);
}

static void asyncCmp(void *argsPtr)
{
  HeTM_thread_s *threadData = (HeTM_thread_s*)argsPtr;
  consecutiveFlagCpy = 0; // allow to cpy the flag
  launchCmpKernel(threadData, threadData->curWSetSize);
}

static void enterBackoffFn()
{
  if (!inBackoff) {
    NVTX_PUSH_RANGE("backoff_mode", NVTX_PROF_BACKOFF);
    inBackoff = 1;
  }
  HeTM_thread_data->statusCMP = HETM_CPY_ASYNC;
  HeTM_thread_data->isCpyDone = 0;
  // CHUNKED_LOG_EXTEND_FORCE(HeTM_thread_data->wSetLog);
  __sync_synchronize(); // sync the log
  HeTM_async_request((HeTM_async_req_s){
    .args = (void*)HeTM_thread_data,
    .fn = asyncCpy
  });

  // TODO: I'm spamming these
  HeTM_async_request((HeTM_async_req_s){
    .args = NULL,
    .fn = asyncGetInterConflFlag
  });
}

static void cpyWSetToGPU()
{
  // did the GPU finished the batch? HeTM_get_GPU_status() is a MACRO
  if (HeTM_get_GPU_status() != HETM_BATCH_DONE
    && HeTM_get_GPU_status() != HETM_GPU_IDLE) return;

  if (HeTM_get_GPU_status() == HETM_GPU_IDLE) {
    // The GPU is IDLE, lets push some writes into the GPU right away
    // continue in case there isn't enough log
    nbCpyRounds = HeTM_thread_data->wSetLog->size / STM_LOG_BUFFER_SIZE;
    if (nbCpyRounds < 1 || !HeTM_thread_data->isCpyDone || !HeTM_thread_data->isCmpDone) return;
  }

  if (!inBackoff) {
    nbCpyRounds = HeTM_thread_data->wSetLog->size / STM_LOG_BUFFER_SIZE;
    TIMER_READ(HeTM_thread_data->backoffBegTimer);
  }

  if (!HeTM_thread_data->isCpyDone && HeTM_thread_data->statusCMP == HETM_CPY_ASYNC) {
    return; // not ready yet
  }
  if (!HeTM_thread_data->isCmpDone && HeTM_thread_data->statusCMP == HETM_CMP_ASYNC) {
    return; // not ready yet
  }

  if (HeTM_thread_data->isCpyDone && HeTM_thread_data->statusCMP == HETM_CPY_ASYNC) {
    HeTM_thread_data->statusCMP = HETM_CMP_ASYNC;
    HeTM_thread_data->isCmpDone = 0;
    HeTM_thread_data->isCmpVoid = 0;

    TIMER_T now;
    TIMER_READ(now);
    HeTM_thread_data->timeCpy = TIMER_DIFF_SECONDS(HeTM_thread_data->beforeCpyLogs, now) * 1000.0f;
    HeTM_thread_data->timeCpySum += HeTM_thread_data->timeCpy;
    HeTM_async_request((HeTM_async_req_s){
      .args = (void*)HeTM_thread_data,
      .fn = asyncCmp
    });

    // FREEs the log used in the transfers
    while (HeTM_thread_data->wSetLog->curr != NULL) {
      chunked_log_node_s *node = HeTM_thread_data->wSetLog->curr;
      HeTM_thread_data->wSetLog->curr = HeTM_thread_data->wSetLog->curr->next;
      CHUNKED_LOG_FREE(node);
    }

    // __sync_synchronize();
    return;
  }

  if (HeTM_thread_data->isCmpDone && HeTM_thread_data->statusCMP == HETM_CMP_ASYNC) { // cmp completed
    HeTM_thread_data->statusCMP = HETM_DONE_ASYNC;
    // TODO: this slows down the GPU --> put into a #ifdef
    if (!HeTM_thread_data->isCmpVoid) {
      CUDA_EVENT_SYNCHRONIZE(HeTM_thread_data->cmpStartEvent);
      CUDA_EVENT_SYNCHRONIZE(HeTM_thread_data->cmpStopEvent);
      CUDA_EVENT_ELAPSED_TIME(&HeTM_thread_data->timeCmp, HeTM_thread_data->cmpStartEvent,
        HeTM_thread_data->cmpStopEvent);
      if (HeTM_thread_data->timeCmp > 0) { // TODO: boggus
        HeTM_thread_data->timeCmpSum += HeTM_thread_data->timeCmp;
      }
    }
    return;
  }

  __sync_synchronize();
	if (HeTM_shared_data.threadsWaitingSync == HeTM_shared_data.nbCPUThreads && doneWithLog) {
    // can only enter here if no cpy or cmp is running
		/* stop sending comparison kernels to the GPU */
    cmpBlockApply();
    // HeTM_sync_barrier(); // /* Wake up GPU controller thread */
    // HeTM_sync_barrier(); // /* wait to set the cuda_stop flag to 0 */
	} else if (HeTM_thread_data->nbCmpLaunches <= nbCpyRounds) {
    // --------------------------------------
    // continue running the CPU
    enterBackoffFn();
    // --------------------------------------
  } else if (!doneWithLog) {
    doneWithLog = 1;
    __sync_add_and_fetch(&HeTM_shared_data.threadsWaitingSync, 1);
    HeTM_thread_data->statusCMP = HETM_CMP_BLOCK;
  }
  HeTM_thread_data->nbCmpLaunches++; // TODO: case that CPU is faster than GPU
}

static void cmpBlockApply()
{
  int i;
#if HETM_LOG_TYPE == HETM_VERS2_LOG
  size_t curNodeSize = !MOD_CHUNKED_LOG_IS_EMPTY(HeTM_thread_data->wSetLog);
#else
  size_t curNodeSize = HeTM_thread_data->wSetLog->size;
#endif

  // TODO: should only be called if CMP_ASYNC before
  HeTM_async_request((HeTM_async_req_s){
    .args = NULL,
    .fn = asyncGetInterConflFlag
  });

//   HETM_DEB_THRD_CPU("Thread %i reachead CMP threshold WSetSize=%zu(x64k)",
//     HeTM_thread_data->id, curNodeSize);
// #if HETM_LOG_TYPE == HETM_VERS2_LOG
//   if (MOD_CHUNKED_LOG_IS_EMPTY(HeTM_thread_data->wSetLog)) {
// #else
//   if (CHUNKED_LOG_IS_EMPTY(HeTM_thread_data->wSetLog)) {
// #endif
//     HeTM_thread_data->isCmpDone = 1;
//     __sync_synchronize();
//   }
  if (inBackoff) {
    // printf("[%i] exit backoff\n", HeTM_thread_data->id);
    NVTX_POP_RANGE();
    inBackoff = 0;
  }
  NVTX_PUSH_RANGE("blocked", NVTX_PROF_BLOCK);
  TIMER_READ(HeTM_thread_data->backoffEndTimer);
  HeTM_thread_data->timeBackoff += TIMER_DIFF_SECONDS(
    HeTM_thread_data->backoffBegTimer, HeTM_thread_data->backoffEndTimer
  );

  if (curNodeSize > 0 && !(HeTM_is_interconflict() && HeTM_shared_data.policy == HETM_CPU_INV)) {
    // must block
    i = 0;
    while (
#if HETM_LOG_TYPE == HETM_VERS2_LOG
      !MOD_CHUNKED_LOG_IS_EMPTY(HeTM_thread_data->wSetLog)
#else
      !CHUNKED_LOG_IS_EMPTY(HeTM_thread_data->wSetLog)
#endif
    ) {
      HeTM_thread_data->isCpyDone = 0;
      HeTM_thread_data->isCmpDone = 0;
      HeTM_thread_data->isCmpVoid = 0;
      // __sync_synchronize();
// #if HETM_LOG_TYPE == HETM_VERS_LOG
//       if (HeTM_thread_data->wSetLog->first->p.pos == 0) break;
// #endif

      __sync_synchronize(); // sync the log
      HeTM_async_request((HeTM_async_req_s){
        .args = (void*)HeTM_thread_data,
        .fn = asyncCpy
      });

      COMPILER_FENCE();
      while (!HeTM_thread_data->isCpyDone) {
        // _mm_pause();
        // pthread_yield(); // block
        // __sync_synchronize();
      }

      TIMER_T now;
      TIMER_READ(now);
      HeTM_thread_data->timeCpy = TIMER_DIFF_SECONDS(HeTM_thread_data->beforeCpyLogs, now) * 1000.0f;
      HeTM_thread_data->timeCpySum += HeTM_thread_data->timeCpy;

      while (HeTM_thread_data->wSetLog->curr != NULL) {
        chunked_log_node_s *node = HeTM_thread_data->wSetLog->curr;
        HeTM_thread_data->wSetLog->curr = HeTM_thread_data->wSetLog->curr->next;
        CHUNKED_LOG_FREE(node);
      }

      // starts the kernel as soon as the memory is copied in that stream
      HeTM_async_request((HeTM_async_req_s){
        .args = (void*)HeTM_thread_data,
        .fn = asyncCmp
      });

      COMPILER_FENCE();
      while (!HeTM_thread_data->isCmpDone) {
        // _mm_pause();
        // pthread_yield(); // block
        // __sync_synchronize();
      }

      if (!HeTM_thread_data->isCmpVoid) {
        CUDA_EVENT_SYNCHRONIZE(HeTM_thread_data->cmpStartEvent);
        CUDA_EVENT_SYNCHRONIZE(HeTM_thread_data->cmpStopEvent);
        CUDA_EVENT_ELAPSED_TIME(&HeTM_thread_data->timeCmp, HeTM_thread_data->cmpStartEvent,
          HeTM_thread_data->cmpStopEvent);
        if (HeTM_thread_data->timeCmp > 0) { // TODO: bug here
          HeTM_thread_data->timeCmpSum += HeTM_thread_data->timeCmp;
        }
      }

      HeTM_async_request((HeTM_async_req_s){
        .args = NULL,
        .fn = asyncGetInterConflFlag
      });

      // wait flag?

      if (HeTM_is_interconflict() && HeTM_shared_data.policy == HETM_CPU_INV) break;
      i++;
    } /* while not empty */
  }  /* no inter-conflict */

  HeTM_sync_barrier(); // /* Wake up GPU controller thread */
  HeTM_sync_barrier(); // /* wait to set the cuda_stop flag to 0 */
  // printf("[%i] <<<<<<<<< NEW ROUND >>>>>>>>>>>>\n", HeTM_thread_data->id);
  NVTX_POP_RANGE();

  HeTM_thread_data->isFirstChunk = 1;

  TIMER_READ(HeTM_thread_data->blockingEndTimer);
  HeTM_thread_data->timeBlocked += TIMER_DIFF_SECONDS(
    HeTM_thread_data->backoffEndTimer, HeTM_thread_data->blockingEndTimer
  );

  // printf("[%i] exit blocked\n", HeTM_thread_data->id);
  HeTM_thread_data->statusCMP = HETM_CMP_OFF;
  HeTM_thread_data->nbCmpLaunches = 0;

  doneWithLog = 0;
  __sync_add_and_fetch(&HeTM_shared_data.threadsWaitingSync, -1);
}

static int launchCmpKernel(HeTM_thread_s *threadData, size_t wsetSize)
{
  HeTM_CPULogEntry *vecDev;
  size_t sizeBuffer = STM_LOG_BUFFER_SIZE * LOG_SIZE;
  int tid = threadData->id;

  vecDev = (HeTM_CPULogEntry*)HeTM_shared_data.wsetLog;
  vecDev += tid*sizeBuffer; // each thread has a bit of the buffer

  TIMER_READ(threadData->beforeCmpKernel);

  // PROBLEM --> the wsetSize is 0 because it was already copied
  if ((HeTM_is_interconflict() && HeTM_shared_data.policy == HETM_CPU_INV) || wsetSize == 0) {
    HETM_DEB_THRD_CPU("Thread %i decided not to CMP (isConfl=%i, wsetSize=%i)",
      threadData->id, HeTM_is_interconflict(), wsetSize);
    threadData->isCmpDone = 1; // TODO: put global
    threadData->isCmpVoid = 1; // TODO: put global
    __sync_synchronize();
    return 0;
  }
  HETM_DEB_THRD_CPU("Thread %i decided to CMP (wsetSize=%i)", threadData->id,
    wsetSize);

#if HETM_CMP_TYPE == HETM_CMP_COMPRESSED
  // -----------------------------------------------
  //Calc number of blocks
  int nbThreadsX = 256;
  int bo = (wsetSize + nbThreadsX-1) / (nbThreadsX);

  // Memory region of the entry object
  HeTM_cmp_s checkTxCompressed_args = {
    .knlArgs = {
      .sizeWSet = (int)wsetSize,
      .sizeRSet = (int)HeTM_shared_data.rsetLogSize,
      .idCPUThr = (int)threadData->id,
    },
    .clbkArgs = threadData
  };

#if HETM_LOG_TYPE == HETM_VERS2_LOG
  // size_t nbGranules = HeTM_shared_data.sizeMemPool / sizeof(PR_GRANULE_T);
  size_t granPerThread = LOG_SIZE*STM_LOG_BUFFER_SIZE;
  nbThreadsX = LOG_THREADS_IN_BLOCK;
  bo = LOG_GPU_THREADS / nbThreadsX;

  // number of entries for each thread
  checkTxCompressed_args.knlArgs.sizeRSet = granPerThread;
#endif /* HETM_LOG_TYPE == HETM_VERS2_LOG */

  dim3 blocksCheck(bo); // partition the stm_log by the different blocks
  dim3 threadsPerBlock(nbThreadsX); // each block has nbThreadsX threads

  // if (wsetSize & 1) {
  //   printf("invalid wsetSize=%i\n", wsetSize);
  // }

  knlman_select("HeTM_checkTxCompressed");
  knlman_set_nb_blocks(bo, 1, 1);
  knlman_set_nb_threads(nbThreadsX, 1, 1);
  knlman_set_entry_object(&checkTxCompressed_args);
  // threadData->didCallCmp = 1;
  knlman_run(threadData->stream);
  // -----------------------------------------------
#elif HETM_CMP_TYPE == HETM_CMP_EXPLICIT
  int xThrs = CMP_EXPLICIT_THRS_PER_RSET / CMP_EXPLICIT_THRS_PER_WSET;
  int yThrs = CMP_EXPLICIT_THRS_PER_WSET;
  int nbGPUThreads = HeTM_shared_data.nbGPUThreads;
  int nbGPUBlocks  = HeTM_shared_data.nbGPUBlocks;
  int blockSize    = HeTM_get_explicit_log_block_size();
  long explicitLogSize = nbGPUThreads * nbGPUBlocks * blockSize;

  int hasRemainderXThrs = explicitLogSize % xThrs;

  int xBlocks = explicitLogSize / xThrs;
  int yBlocks = wsetSize / CMP_EXPLICIT_THRS_PER_WSET;

  int hasRemainderYBlocks = wsetSize % CMP_EXPLICIT_THRS_PER_WSET;

  if (hasRemainderXThrs) xBlocks++;
  if (hasRemainderYBlocks) yBlocks++;

  knlman_select("HeTM_checkTxExplicit");
  knlman_set_nb_blocks(xBlocks, yBlocks, 1);
  knlman_set_nb_threads(xThrs, yThrs, 1);

  HeTM_cmp_s checkTxExplicit_args = {
    .knlArgs = {
      .sizeWSet = (int)wsetSize,
      .sizeRSet = (int)explicitLogSize,
    },
    .clbkArgs = threadData
  };
  knlman_set_entry_object(&checkTxExplicit_args);
  knlman_run(threadData->stream);
#else
  printf("Error! no compare method selected!\n");
#endif

  return 0;
}

static void checkCmpDone()
{
  if (HeTM_thread_data->isCmpDone) {
    // No limit for the number of rounds
    if (HeTM_is_interconflict() && HeTM_shared_data.policy == HETM_CPU_INV) {
      if (!doneWithLog) {
        doneWithLog = 1;
        __sync_add_and_fetch(&HeTM_shared_data.threadsWaitingSync, 1);
      }
      HeTM_thread_data->statusCMP = HETM_CMP_BLOCK;
      __sync_synchronize();
    }
  }
}

static void asyncGetInterConflFlag(void*)
{
  if (!consecutiveFlagCpy) {
    consecutiveFlagCpy = 1; // TODO: memory barrier missing
    HeTM_set_is_interconflict(HeTM_get_inter_confl_flag(HeTM_memStream));
  }
}
#endif /* HETM_LOG_TYPE != HETM_BMAP_LOG */
