#include "hip/hip_runtime.h"
#include "hetm.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <errno.h>
#include <string.h>
#include <hiprand/hiprand_kernel.h>
#include "memman.h"
#include "knlman.h"
#include "hetm-log.h"
#include "pr-stm-wrapper.cuh"
#include "hetm-cmp-kernels.cuh"

#include <map>

using namespace std;

static map<void*, size_t> alloced;
static map<void*, size_t> freed;
static size_t curSize;

static size_t bitmapGran = DEFAULT_BITMAP_GRANULARITY;
// static size_t bitmapGranBits = DEFAULT_BITMAP_GRANULARITY_BITS;

#if HETM_LOG_TYPE != HETM_BMAP_LOG
static void CUDART_CB cpyCallback(hipStream_t event, hipError_t status, void *data);
#endif

static void init_mempool(size_t pool_size);
static void init_RSetWSet(size_t pool_size);
static void init_interConflFlag();

#if HETM_LOG_TYPE == HETM_VERS_LOG
  static void init_vers(size_t pool_size);
#elif  HETM_LOG_TYPE == HETM_VERS2_LOG
  static void init_vers2(size_t pool_size);
#elif  HETM_LOG_TYPE == HETM_ADDR_LOG
  static void init_addr(size_t pool_size);
#elif  HETM_LOG_TYPE == HETM_BMAP_LOG
  static void init_bmap(size_t pool_size);
#endif

// TODO: this function is too long --> chunk it!
int HeTM_mempool_init(size_t pool_size)
{
  size_t nbGranules = pool_size / PR_LOCK_GRANULARITY;
  size_t nbChunks;

  init_mempool(pool_size);

#if HETM_LOG_TYPE == HETM_VERS_LOG
  init_vers(pool_size);
#elif  HETM_LOG_TYPE == HETM_VERS2_LOG
  init_vers2(pool_size);
#elif  HETM_LOG_TYPE == HETM_ADDR_LOG
  init_addr(pool_size);
#elif  HETM_LOG_TYPE == HETM_BMAP_LOG
  init_bmap(pool_size);
#else /* HETM_LOG_TYPE not defined */
  printf(" >>> ERROR! HETM_LOG_TYPE not defined\n");
  exit(EXIT_FAILURE);
#endif

  init_interConflFlag();
  init_RSetWSet(pool_size);

  nbChunks = pool_size / DEFAULT_BITMAP_GRANULARITY;
  if (nbGranules % DEFAULT_BITMAP_GRANULARITY > 0) nbChunks++;
  HeTM_shared_data.nbChunks = nbChunks;

  // printf(" <<<<<<< HeTM_shared_data.devMemPoolBackupBmap = %p\n",  HeTM_shared_data.devMemPoolBackupBmap);

  HeTM_set_global_arg((HeTM_knl_global_s){
    .devMemPoolBasePtr  = HeTM_shared_data.devMemPool,
// #if HETM_LOG_TYPE == HETM_ADDR_LOG || HETM_LOG_TYPE == HETM_BMAP_LOG
    // .devMemPoolBackupBasePtr = HeTM_shared_data.devMemPoolBackup,
    .devMemPoolBackupBasePtr = HeTM_shared_data.devMemPoolBackup,
    .devMemPoolBackupBmap = ((memman_bmap_s*)HeTM_shared_data.devMemPoolBackupBmap)->dev,
// #endif
    .hostMemPoolBasePtr = HeTM_shared_data.hostMemPool,
    .versions           = HeTM_shared_data.devVersions,
    .isInterConfl       = HeTM_shared_data.devInterConflFlag,
    .explicitLogBlock   = HeTM_get_explicit_log_block_size(),
    .nbGranules         = nbGranules, // TODO: granules
    .devRSet            = HeTM_shared_data.rsetLog,
    .hostWSet           = HeTM_shared_data.wsetLog,
    .hostWSetCache      = HeTM_shared_data.wsetCache,
    .hostWSetCacheConfl = HeTM_shared_data.wsetCacheConfl,
    .hostWSetCacheSize  = HeTM_shared_data.wsetCacheSize,
    .hostWSetCacheBits  = HeTM_shared_data.wsetCacheBits,
    .hostWSetChunks     = nbChunks,
    .PRLockTable        = PR_lockTableDev,
    .randState          = HeTM_shared_data.devCurandState,
    .isGPUOnly          = (HeTM_shared_data.isCPUEnabled == 0)
  });

  curSize = 0;
  return 0; // TODO: check for error
}

int HeTM_mempool_destroy()
{
  memman_select("HeTM_mempool");
  memman_free_dual();
  if(!memman_select("HeTM_gpuLog"))
    memman_free_dual();
  memman_select("HeTM_versions");
  memman_free_gpu();
  memman_select("HeTM_dev_rset");
  memman_free_gpu();
  memman_select("HeTM_host_wset");
  memman_free_gpu();
  return 0;
}

void HeTM_initCurandState()
{
  int nbThreads = HeTM_shared_data.nbGPUThreads;
  int nbBlocks = HeTM_shared_data.nbGPUBlocks;
  size_t size = nbThreads * nbBlocks * sizeof(long); // TODO: from sizeof(hiprandState)
  memman_alloc_gpu("HeTM_curand_state", size, NULL, 0);
  HeTM_shared_data.devCurandState = memman_get_gpu(NULL);
  HeTM_setupCurand<<<nbBlocks, nbThreads>>>(HeTM_shared_data.devCurandState);
  hipDeviceSynchronize(); // TODO: blocks
}

void HeTM_destroyCurandState()
{
  memman_select("HeTM_curand_state");
  memman_free_gpu();
}

int HeTM_mempool_cpy_to_cpu(size_t *copiedSize)
{
#ifndef USE_UNIF_MEM
  memman_select("HeTM_mempool");
  memman_cpy_to_cpu(HeTM_memStream, copiedSize);
  return 0; // TODO: error code
#endif /* USE_UNIF_MEM */
}

int HeTM_mempool_cpy_to_gpu(size_t *copiedSize)
{
#ifndef USE_UNIF_MEM
  memman_select("HeTM_mempool");
  memman_cpy_to_gpu(HeTM_memStream, copiedSize);
#endif /* USE_UNIF_MEM */
  return 0; // TODO: error code
}

#if HETM_LOG_TYPE == HETM_VERS2_LOG
void HeTM_wset_log_cpy_to_gpu(
  HeTM_thread_s *threadData, chunked_log_node_s *node, size_t *size
) {
  char *buffer = node->chunk;
  hipStream_t stream = (hipStream_t)threadData->stream;
  int id = threadData->id;
  size_t emptySpace;
  size_t sizeOneChunk = STM_LOG_BUFFER_SIZE*LOG_SIZE*LOG_ACTUAL_SIZE*sizeof(HeTM_CPULogEntry);

  void *ptrGPUBuffer = ((char*)HeTM_shared_data.wsetLog) + id * sizeOneChunk;

  threadData->countCpy = 1; // TODO: start with big number
  threadData->nbCpyDone = 0;
  __sync_synchronize();
  CUDA_EVENT_RECORD(threadData->cpyWSetStartEvent, stream);
  // hipMemset(ptrGPUBuffer, 0, sizeOneChunk); // sync

  // int i;
  // for (i = 0; i < STM_LOG_BUFFER_SIZE*LOG_SIZE*LOG_ACTUAL_SIZE; ++i) {
  //   HeTM_CPULogEntry *entry = (HeTM_CPULogEntry*)(&buffer[i*sizeof(HeTM_CPULogEntry)]);
  //   if (entry->pos != 0 && ((char*)entry->pos < (char*)HeTM_shared_data.hostMemPool ||
  //       (char*)entry->pos > (char*)HeTM_shared_data.hostMemPool + HeTM_shared_data.sizeMemPool)) {
  //     printf("wrong addr, i=%i, ptr=%p\n", i, entry->pos);
  //   }
  // }

  CUDA_CPY_TO_DEV_ASYNC(ptrGPUBuffer, buffer, sizeOneChunk, stream);
  HeTM_stats_data.sizeCpyLogs += sizeOneChunk;
  // emptySpace = MOD_CHUNKED_LOG_NODE_FREE_SPACE(node);
  // HeTM_stats_data.sizeCpyWSet += emptySpace;
  if (threadData->isFirstChunk) {
    threadData->emptySpaceFirstChunk += emptySpace;
    threadData->isFirstChunk = 0;
  }
  CUDA_CHECK_ERROR(hipStreamAddCallback(
    stream, cpyCallback, threadData, 0
  ), "");
  COMPILER_FENCE();
  if (threadData->nbCpyDone >= threadData->countCpy) {
    threadData->isCpyDone = 1;
  }
  __sync_synchronize();
  CUDA_EVENT_RECORD(threadData->cpyWSetStopEvent, stream);
}

#elif HETM_LOG_TYPE != HETM_BMAP_LOG

void HeTM_wset_log_cpy_to_gpu(
  HeTM_thread_s *threadData, chunked_log_node_s *node, size_t *size
) {
  void *res;
  HeTM_CPULogEntry *resAux;
  size_t sizeRes = 0, sizeToCpy, sizeBuffer, nbEntries;
  chunked_log_node_s *logAux;
  int tid = threadData->id;
  hipStream_t stream = (hipStream_t)threadData->stream;
  int count = 0;

  sizeBuffer = STM_LOG_BUFFER_SIZE * LOG_SIZE;

  res = HeTM_shared_data.wsetLog;
  resAux = (HeTM_CPULogEntry*)res;
  resAux += tid*sizeBuffer; // each thread has a bit of the buffer
  // TODO: this memset is needed
  // CUDA_CHECK_ERROR(hipMemsetAsync(resAux, 0, sizeBuffer*sizeof(HeTM_CPULogEntry)), "");

  logAux = node;
  threadData->countCpy = 1000; // TODO: start with big number
  threadData->nbCpyDone = 0;
  TIMER_READ(threadData->beforeCpyLogs);
  // CUDA_EVENT_RECORD(threadData->cpyWSetStartEvent, stream); // USE TIMER_T
  size_t totCpy = 0;
  while (logAux != NULL) {
    sizeToCpy = logAux->p.pos; // size in bytes
    nbEntries = logAux->p.pos/sizeof(HeTM_CPULogEntry);
    if (sizeToCpy <= 0 || nbEntries > sizeBuffer || logAux->chunk == NULL) {
      // TODO: logAux->chunk == NULL is a big bug
      // printf("INVALID SIZE %zu!!! %p --> %p nextLog=%p\n", sizeToCpy, resAux, logAux->chunk, logAux->next); // TODO
      node = logAux;
      logAux = logAux->next;
      // CHUNKED_LOG_FREE(node); // TODO: target thread must free
      continue;
    }
    if (CUDA_CPY_TO_DEV_ASYNC(resAux, logAux->chunk, sizeToCpy, stream) != hipSuccess) {
      printf("ERROR copying Dev %p/%p <-- %p/%p Host (nbEntries=%zu) \n"
             "           BUFFER=%p/%p totalCpy=%zu maxSize=%zu\n",
        resAux, ((char*)resAux)+sizeToCpy, logAux->chunk, logAux->chunk+sizeToCpy, nbEntries, HeTM_shared_data.wsetLog,
        ((HeTM_CPULogEntry*)HeTM_shared_data.wsetLog) + HeTM_shared_data.wsetLogSize,
        totCpy, sizeBuffer*sizeof(HeTM_CPULogEntry));
    }
    totCpy += sizeToCpy;
    count++;

    resAux += nbEntries; // move ahead of the copied position
    sizeRes += nbEntries;

    // TODO: now the log is <= STM_LOG_BUFFER_SIZE

    node = logAux;
    logAux = logAux->next;
    // CHUNKED_LOG_FREE(node); // TODO: target thread must free

    if (totCpy >= sizeBuffer*sizeof(HeTM_CPULogEntry)) {
      // no more space in the buffer try latter
      break;
    }
  }
  if (totCpy > 0) {
    CUDA_CHECK_ERROR(hipStreamAddCallback(
      stream, cpyCallback, threadData, 0
    ), "");
  } else {
    threadData->isCpyDone = 1;
  }

  HeTM_stats_data.sizeCpyLogs += totCpy;
  // HeTM_stats_data.sizeCpyWSet += sizeBuffer - totCpy; // empty space
  __sync_synchronize();

  if (size != NULL) *size = sizeRes; // returns the number of copied entries
  // CUDA_EVENT_RECORD(threadData->cpyWSetStopEvent, stream); // USE TIMER_T
}
#endif /* HETM_LOG_TYPE != HETM_BMAP_LOG */

int HeTM_alloc(void **cpu_ptr, void **gpu_ptr, size_t size)
{
  size_t newSize = curSize + size;
  if (newSize > HeTM_shared_data.sizeMemPool) {
    // TODO: check the freed memory
    return -1;
  }

  // there is still space left
  char *curPtrHost = (char*)HeTM_shared_data.hostMemPool;
  char *curPtrDev  = (char*)HeTM_shared_data.devMemPool;
  curPtrHost += curSize;
  curPtrDev  += curSize;
  if (cpu_ptr) *cpu_ptr = (void*)curPtrHost;
  if (gpu_ptr) *gpu_ptr = (void*)curPtrDev;
  curSize = newSize;
  alloced.insert(make_pair(*cpu_ptr, size));

  return 0;
}

int HeTM_free(void **cpu_ptr)
{
  // TODO:
  auto it = alloced.find(*cpu_ptr);
  if (it == alloced.end()) {
    return -1; // not found
  }
  freed.insert(make_pair(*cpu_ptr, it->second));
  alloced.erase(it);
  return 0;
}

void* HeTM_map_addr_to_gpu(void *origin)
{
  uintptr_t o = (uintptr_t)origin;
  uintptr_t host = (uintptr_t)HeTM_shared_data.hostMemPool;
  uintptr_t dev  = (uintptr_t)HeTM_shared_data.devMemPool;
  return (void*)(o - host + dev);
}

void* HeTM_map_cpu_to_cpu(void *origin)
{
  uintptr_t o = (uintptr_t)origin;
  uintptr_t host = (uintptr_t)HeTM_shared_data.hostMemPool;
  uintptr_t dev  = (uintptr_t)HeTM_shared_data.devMemPool;
  return (void*)(o - dev + host);
}

int HeTM_reset_GPU_state()
{
  // CUDA_CHECK_ERROR(hipMemset(PR_lockTableDev, 0, PR_LOCK_TABLE_SIZE*sizeof(int)), "");
  HeTM_reset_inter_confl_flag();
  memman_select("HeTM_dev_rset");
  memman_zero_gpu(NULL);
  return 0;
}

#if HETM_LOG_TYPE != HETM_BMAP_LOG
static void CUDART_CB cpyCallback(hipStream_t event, hipError_t status, void *data)
{
  HeTM_thread_s *threadData = (HeTM_thread_s*)data;

  if(status != hipSuccess) { // TODO: Handle error
    printf("CMP failed! >>> %s <<<\n", hipGetErrorString(status));
    // TODO: exit application
  }

  // threadData->nbCpyDone++;
  // if (threadData->nbCpyDone >= threadData->countCpy) {
    threadData->isCpyDone = 1;
  //   // TODO: erase values?
  // }

  TIMER_T now;
  TIMER_READ(now);
  double timeTaken = TIMER_DIFF_SECONDS(threadData->beforeCpyLogs, now);
  // printf(" --- send to %i wait COPY delay=%fms\n", threadData->id, timeTaken*1000);
  threadData->timeLogs += timeTaken;

  __sync_synchronize(); // cmpCallback is called from a different thread
}
#endif

static void init_mempool(size_t pool_size)
{
  size_t granBmap;
#ifdef USE_UNIF_MEM
  memman_alloc_dual("HeTM_mempool", pool_size, MEMMAN_UNIF);
#else /* !USE_UNIF_MEM */
  memman_alloc_dual("HeTM_mempool", pool_size, MEMMAN_NONE);
#endif /* USE_UNIF_MEM */
  HeTM_shared_data.devMemPool  = memman_get_gpu(NULL);
  HeTM_shared_data.hostMemPool = memman_get_cpu(NULL);
  HeTM_shared_data.sizeMemPool = pool_size;
  stm_baseMemPool = HeTM_shared_data.hostMemPool;

  // bmap for the data-set chunking
  // TODO: bitmap is x4B larger (does not take in account ints of 4B)
  memman_create_bitmap(HeTM_shared_data.hostMemPool, HeTM_shared_data.devMemPool, bitmapGran);
  memman_bitmap_gpu(); // creates in GPU (TODO: change name)
  memman_bmap_s *mainBMap = (memman_bmap_s*) memman_get_bmap(&granBmap);
  stm_devMemPoolBmap = mainBMap;

#ifdef HETM_DISABLE_CHUNKS // TODO: ADD YET ANOTHER FLAG!!!
  memman_set_is_bmapped(0, 0);
  memman_alloc_gpu("HeTM_mempool_backup", pool_size,
    HeTM_shared_data.hostMemPool, MEMMAN_NONE);
  HeTM_shared_data.devMemPoolBackup = memman_get_gpu(NULL);
  HeTM_shared_data.devMemPoolBackupBmap = memman_get_bmap(NULL);
  stm_devMemPoolBackupBmap = NULL;
#else /* HETM_CHUNKS enabled */
  memman_set_is_bmapped(1, 1);
  memman_alloc_gpu("HeTM_mempool_backup", pool_size,
    HeTM_shared_data.hostMemPool, MEMMAN_NONE);
  memman_create_bitmap(memman_get_cpu(NULL), memman_get_gpu(NULL), bitmapGran);
  memman_bitmap_gpu(); // creates in GPU (TODO: change name)
  memman_bmap_s *backupBMap = (memman_bmap_s*) memman_get_bmap(NULL);
  stm_devMemPoolBackupBmap = backupBMap;

  memman_attach_bmap(stm_devMemPoolBackupBmap, granBmap);
  memman_set_is_bmapped(1, 1);
  HeTM_shared_data.devMemPoolBackup = memman_get_gpu(NULL);
  HeTM_shared_data.devMemPoolBackupBmap = memman_get_bmap(NULL);
#endif /* HETM_DISABLE_CHUNKS */

  // memman_select("HeTM_mempool");
  // memman_bmap_s *mainBMap = (memman_bmap_s*) memman_get_bmap(NULL);

  // TODO: what is this doing?
  memman_alloc_gpu("HeTM_mempool_backup_bmap", sizeof(memman_bmap_s), backupBMap, MEMMAN_NONE);
  memman_alloc_gpu("HeTM_mempool_bmap", sizeof(memman_bmap_s), mainBMap, MEMMAN_NONE);
}

static void init_interConflFlag() {
  memman_alloc_dual("HeTM_interConflFlag", sizeof(int), MEMMAN_NONE);
  HeTM_shared_data.hostInterConflFlag = (int*)memman_get_cpu(NULL);
  HeTM_shared_data.devInterConflFlag  = (int*)memman_get_gpu(NULL);
}

static void init_RSetWSet(size_t pool_size) {
  size_t sizeRSetLog = 0;

#if HETM_CMP_TYPE == HETM_CMP_EXPLICIT
  sizeRSetLog = HeTM_shared_data.nbGPUThreads*HeTM_shared_data.nbGPUBlocks
    *HeTM_get_explicit_log_block_size()*sizeof(PR_GRANULE_T);
#elif HETM_CMP_TYPE == HETM_CMP_COMPRESSED
  // Bitmap with 1 byte per account, 1 means accessed (TODO: /8 + atomicOr)
  sizeRSetLog = pool_size / PR_LOCK_GRANULARITY;
#else
    // Error or disabled
#endif

  if (sizeRSetLog > 0) {
    // Inits GPU read-set log
    memman_alloc_gpu("HeTM_dev_rset", sizeRSetLog, NULL, 0);
    memman_zero_gpu(NULL);
    HeTM_shared_data.rsetLog = memman_get_gpu(NULL);
    HeTM_shared_data.rsetLogSize = sizeRSetLog;

    // inits CPU write-set buffer
#if HETM_LOG_TYPE != HETM_BMAP_LOG
    size_t sizeWSetBuffer = 0;
    // TODO: using the bitmap in the cpu side
    sizeWSetBuffer = STM_LOG_BUFFER_SIZE*LOG_SIZE*sizeof(HeTM_CPULogEntry);
#if HETM_LOG_TYPE == HETM_VERS2_LOG
    sizeWSetBuffer *= LOG_ACTUAL_SIZE;
#endif
    sizeWSetBuffer *= HeTM_shared_data.nbCPUThreads; // 1 buffer per thread
    memman_alloc_gpu("HeTM_host_wset", sizeWSetBuffer, NULL, 0);
    memman_zero_gpu(NULL);
    HeTM_shared_data.wsetLog = memman_get_gpu(NULL);
    HeTM_shared_data.wsetLogSize = sizeWSetBuffer;
#endif
  } // else CMP is disabled
}

#if HETM_LOG_TYPE == HETM_BMAP_LOG
static void init_bmap(size_t pool_size)
{
  size_t nbGranules = pool_size / PR_LOCK_GRANULARITY;
  size_t granBmap;

  size_t cacheSize = pool_size / CACHE_GRANULE_SIZE; // nbGranules / CACHE_GRANULE_SIZE;

  if (pool_size % CACHE_GRANULE_SIZE > 0) {
    cacheSize++;
  }

  memman_alloc_dual("HeTM_cpu_wset_cache", cacheSize, MEMMAN_NONE);

  // GPU set to 1 to say there was a conflict
  memman_alloc_dual("HeTM_cpu_wset_cache_confl", cacheSize, MEMMAN_NONE);

  memman_select("HeTM_mempool");
  memman_bmap_s *mainBMap = (memman_bmap_s*) memman_get_bmap(&granBmap);

#ifdef HETM_DISABLE_CHUNKS
  // memman_alloc_gpu("HeTM_mempool_backup", pool_size,
  //   HeTM_shared_data.hostMemPool, MEMMAN_NONE);
  // HeTM_shared_data.devMemPoolBackup = memman_get_gpu(NULL);
  // stm_devMemPoolBackupBmap = NULL;
  memman_alloc_dual("HeTM_cpu_wset", nbGranules, MEMMAN_NONE);
#else /* HETM CHUNKS ENABLED */
  // memman_alloc_gpu("HeTM_mempool_backup", pool_size,
  //   HeTM_shared_data.hostMemPool, MEMMAN_NONE);
  // // memman_create_bitmap(memman_get_cpu(NULL), NULL, bitmapGran);
  // stm_devMemPoolBackupBmap = mainBMap;
  // memman_attach_bmap(stm_devMemPoolBackupBmap, granBmap);
  // memman_set_is_bmapped(1, 0);
  // HeTM_shared_data.devMemPoolBackup = memman_get_gpu(NULL);
  memman_alloc_dual("HeTM_cpu_wset", nbGranules, MEMMAN_NONE);
  memman_attach_bmap(stm_devMemPoolBackupBmap, granBmap / 4); /* 1B maps 4B */
  memman_set_is_bmapped(1, 0); // TODO: this is not working anymore...
#endif /* HETM_DISABLE_CHUNKS */

  memman_zero_cpu(NULL);
  memman_zero_gpu(NULL);
  HeTM_shared_data.wsetLogSize = nbGranules;
  HeTM_shared_data.wsetLog     = memman_get_gpu(NULL);
  stm_wsetCPU                  = memman_get_cpu(NULL);

  memman_select("HeTM_cpu_wset_cache");
  memman_zero_cpu(NULL);
  memman_zero_gpu(NULL);
  HeTM_shared_data.wsetCache     = memman_get_gpu(NULL);
  HeTM_shared_data.wsetCacheSize = CACHE_GRANULE_SIZE;
  HeTM_shared_data.wsetCacheBits = CACHE_GRANULE_BITS;
  stm_wsetCPUCache               = memman_get_cpu(NULL);
  stm_wsetCPUCacheBits           = CACHE_GRANULE_BITS;

  memman_select("HeTM_cpu_wset_cache_confl");
  memman_zero_cpu(NULL);
  memman_zero_gpu(NULL);
  HeTM_shared_data.wsetCacheConfl = memman_get_gpu(NULL);

  memman_select("HeTM_mempool");
  HeTM_shared_data.hostMemPool = memman_get_cpu(NULL);
  // HeTM_shared_data.devMemPool  = memman_get_gpu(NULL);
  HeTM_shared_data.sizeMemPool = pool_size;
  stm_baseMemPool = HeTM_shared_data.hostMemPool;
}
#endif /* HETM_LOG_TYPE == HETM_BMAP_LOG */

#if HETM_LOG_TYPE == HETM_ADDR_LOG
static void init_addr(size_t pool_size)
{
  size_t nbGranules = pool_size / PR_LOCK_GRANULARITY;
  size_t granBmap;

  // Inits mempool TODO!!! --> GPU also chunked
  memman_alloc_dual("HeTM_mempool", pool_size, MEMMAN_NONE); // TODO: trade-offs with MEMMAN_UNIF
  HeTM_shared_data.devMemPool  = memman_get_gpu(NULL);

  // bmap for the data-set chunking
  memman_create_bitmap(memman_get_cpu(NULL), memman_get_gpu(NULL), bitmapGran);
  memman_bitmap_gpu(); // creates in GPU (TODO: change name)
#ifdef HETM_DISABLE_CHUNKS // TODO: ADD YET ANOTHER FLAG!!!
  memman_set_is_bmapped(0, 0);
#else /* HETM_CHUNKS enabled */
  memman_set_is_bmapped(0, 1);
#endif /* HETM_DISABLE_CHUNKS */
  memman_bmap_s *mainBMap = (memman_bmap_s*) memman_get_bmap(NULL);
  memman_alloc_gpu("HeTM_mempool_bmap", sizeof(memman_bmap_s), mainBMap, MEMMAN_NONE);

  memman_select("HeTM_mempool");
  HeTM_shared_data.hostMemPool = memman_get_cpu(NULL);
  // TODO: too many combinations
#ifdef HETM_DISABLE_CHUNKS
  // memman_alloc_gpu("HeTM_mempool_backup", pool_size,
  //   HeTM_shared_data.hostMemPool, MEMMAN_NONE);
  // stm_devMemPoolBackupBmap = NULL;
#else /* HETM CHUNKS ENABLED */
  // memman_alloc_gpu("HeTM_mempool_backup", pool_size,
  //   HeTM_shared_data.hostMemPool, MEMMAN_NONE);
  // memman_create_bitmap(memman_get_cpu(NULL), NULL, bitmapGran);
  // memman_set_is_bmapped(1, 0);
  // stm_devMemPoolBackupBmap = memman_get_bmap(&granBmap);
#endif /* HETM_DISABLE_CHUNKS */
  HeTM_shared_data.devMemPoolBackup = memman_get_gpu(NULL);

  memman_select("HeTM_mempool");
  HeTM_shared_data.hostMemPool = memman_get_cpu(NULL);
  // HeTM_shared_data.devMemPool  = memman_get_gpu(NULL);
  HeTM_shared_data.sizeMemPool = pool_size;
  stm_baseMemPool = HeTM_shared_data.hostMemPool;

  // Inits versions buffer
  // TODO: the versions array occupies more space than the dataset!!!

  // TODO: I'm using the HeTM_versions in HETM_ADDR_LOG as CPU-WSet
  size_t sizeVersion = nbGranules*sizeof(char); // WSet bmap
  memman_alloc_gpu("HeTM_versions", sizeVersion, NULL, MEMMAN_NONE);
  HeTM_shared_data.devVersions = memman_get_gpu(NULL);
  memman_zero_gpu(NULL);

  // Inits inter-conflict flag
  memman_alloc_dual("HeTM_interConflFlag", sizeof(int), MEMMAN_NONE);
  HeTM_shared_data.hostInterConflFlag = (int*)memman_get_cpu(NULL);
  HeTM_shared_data.devInterConflFlag  = (int*)memman_get_gpu(NULL);
}
#endif /* HETM_LOG_TYPE == HETM_ADDR_LOG */

#if HETM_LOG_TYPE == HETM_VERS_LOG
static void init_vers(size_t pool_size) {
  size_t nbGranules = pool_size / PR_LOCK_GRANULARITY;

  memman_select("HeTM_mempool");
  HeTM_shared_data.hostMemPool = memman_get_cpu(NULL);
  // HeTM_shared_data.devMemPool  = memman_get_gpu(NULL);
  HeTM_shared_data.sizeMemPool = pool_size;
  stm_baseMemPool = HeTM_shared_data.hostMemPool;

  // Inits versions buffer
  // TODO: the versions array occupies more space than the dataset!!!

  size_t sizeVersion = nbGranules*sizeof(long);
  memman_alloc_gpu("HeTM_versions", sizeVersion, NULL, MEMMAN_NONE);
  HeTM_shared_data.devVersions = memman_get_gpu(NULL);
  memman_zero_gpu(NULL);
}
#endif /* HETM_LOG_TYPE == HETM_VERS_LOG */

#if HETM_LOG_TYPE == HETM_VERS2_LOG
static void init_vers2(size_t pool_size) {
  size_t nbGranules = pool_size / PR_LOCK_GRANULARITY;
  size_t sizeRSetLog = 0;
  size_t sizeWSetBuffer = 0;

  // Inits mempool TODO!!! --> GPU also chunked
  memman_alloc_dual("HeTM_mempool", pool_size, MEMMAN_NONE); // TODO: trade-offs with MEMMAN_UNIF
  HeTM_shared_data.devMemPool  = memman_get_gpu(NULL);

  // bmap for the data-set chunking
  memman_create_bitmap(memman_get_cpu(NULL), memman_get_gpu(NULL), bitmapGran);
  memman_bitmap_gpu(); // creates in GPU (TODO: change name)
#ifdef HETM_DISABLE_CHUNKS // TODO: ADD YET ANOTHER FLAG!!!
  memman_set_is_bmapped(0, 0);
#else /* HETM_CHUNKS enabled */
  memman_set_is_bmapped(0, 1);
#endif /* HETM_DISABLE_CHUNKS */
  memman_bmap_s *mainBMap = (memman_bmap_s*) memman_get_bmap(NULL);
  memman_alloc_gpu("HeTM_mempool_bmap", sizeof(memman_bmap_s), mainBMap, MEMMAN_NONE);

  memman_select("HeTM_mempool");
  HeTM_shared_data.hostMemPool = memman_get_cpu(NULL);
  // HeTM_shared_data.devMemPool  = memman_get_gpu(NULL);
  HeTM_shared_data.sizeMemPool = pool_size;
  stm_baseMemPool = HeTM_shared_data.hostMemPool;

  // Inits versions buffer
  // TODO: the versions array occupies more space than the dataset!!!

  size_t sizeVersion = nbGranules*sizeof(long);
  memman_alloc_gpu("HeTM_versions", sizeVersion, NULL, MEMMAN_NONE);
  memman_select("HeTM_versions");
  HeTM_shared_data.devVersions = memman_get_gpu(NULL);
  memman_zero_gpu(NULL);
}
#endif /* HETM_LOG_TYPE == HETM_VERS2_LOG */
